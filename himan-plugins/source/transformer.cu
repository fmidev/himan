#include "hip/hip_runtime.h"
#include "cuda_plugin_helper.h"

using namespace himan;

template <typename T>
__global__ void TransformerKernel(const T* __restrict__ d_source, T* __restrict__ d_dest, double scale, double base,
                                  size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		d_dest[idx] = __fma_rn(d_source[idx], scale, base);
	}
}

namespace transformergpu
{
void Process(std::shared_ptr<const himan::plugin_configuration> conf, std::shared_ptr<info<double>> myTargetInfo,
             std::shared_ptr<info<double>> sourceInfo, double scale, double base)
{
	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	const size_t N = myTargetInfo->SizeLocations();
	size_t memsize = N * sizeof(double);

	// Allocate device arrays

	double *d_source = 0, *d_dest = 0;

	// Allocate memory on device

	CUDA_CHECK(hipMalloc((void**)&d_source, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_dest, memsize));

	// Copy data to device

	cuda::PrepareInfo(sourceInfo, d_source, stream, conf->UseCacheForReads());

	// dims

	const int blockSize = 512;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	CUDA_CHECK(hipStreamSynchronize(stream));

	TransformerKernel<double><<<gridSize, blockSize, 0, stream>>>(d_source, d_dest, scale, base, N);
	cuda::ReleaseInfo(myTargetInfo, d_dest, stream);

	// block until the stream has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_source));
	CUDA_CHECK(hipFree(d_dest));

	hipStreamDestroy(stream);
}
}
