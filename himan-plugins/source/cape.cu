#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/execution_policy.h>

#include "plugin_factory.h"

#include "cape.cuh"
#include "cuda_helper.h"
#include "lift.h"
#include "util.h"

#include "cuda_plugin_helper.h"
#include "forecast_time.h"
#include "level.h"
#include "timer.h"

#define HIMAN_AUXILIARY_INCLUDE

#include "hitool.h"

#include "debug.h"

#undef HIMAN_AUXILIARY_INCLUDE

using namespace himan;
using namespace himan::numerical_functions;
using namespace himan::plugin;

himan::level cape_cuda::itsBottomLevel;
bool cape_cuda::itsUseVirtualTemperature;

typedef std::vector<std::vector<float>> vec2d;

extern float Max(const std::vector<float>& vec);
extern std::tuple<vec2d, vec2d, vec2d> GetSampledSourceData(std::shared_ptr<const plugin_configuration> conf,
                                                            std::shared_ptr<info<float>> myTargetInfo,
                                                            const std::vector<float>& P500m,
                                                            const std::vector<float>& Psurface, const level& startLevel,
                                                            const level& stopLevel);

template <typename T>
__global__ void InitializeArrayKernel(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = val;
	}
}

template <typename T>
void InitializeArray(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	InitializeArrayKernel<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

template <typename T>
__global__ void MultiplyWith(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = d_arr[idx] * val;
	}
}

template <typename T>
void MultiplyWith(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	MultiplyWith<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

__global__ void LastLFCCopyKernel(const float* __restrict__ d_LFCT, const float* __restrict__ d_LFCP,
                                  float* __restrict__ d_LastLFCT, float* __restrict__ d_LastLFCP, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		if (IsMissing(d_LastLFCT[idx]) || d_LastLFCP[idx] > d_LFCP[idx])
		{
			d_LastLFCT[idx] = d_LFCT[idx];
			d_LastLFCP[idx] = d_LFCP[idx];
		}
	}
}

__global__ void CapELValuesKernel(const float* __restrict__ d_CAPE, float* __restrict__ d_ELT,
                                  float* __restrict__ d_ELP, float* __restrict__ d_ELZ, float* __restrict__ d_LastELT,
                                  float* __restrict__ d_LastELP, float* __restrict__ d_LastELZ,
                                  const float* __restrict__ d_Tenv, const float* __restrict__ d_Penv,
                                  const float* __restrict__ d_Zenv, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		// If the CAPE area is continued all the way to stopLevel and beyond, we don't have an EL for that
		// (since integration is forcefully stopped)
		// In this case let last level be EL

		if (d_CAPE[idx] > 0 && IsMissing(d_ELT[idx]))
		{
			d_ELT[idx] = d_Tenv[idx];
			d_ELP[idx] = d_Penv[idx];
			d_ELZ[idx] = d_Zenv[idx];

			d_LastELT[idx] = d_Tenv[idx];
			d_LastELP[idx] = d_Penv[idx];
			d_LastELZ[idx] = d_Zenv[idx];
		}
	}
}

__global__ void VirtualTemperatureKernel(float* __restrict__ d_T, const float* __restrict__ d_P, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		d_T[idx] = himan::metutil::VirtualTemperature_<float>(d_T[idx], d_P[idx] * 100);
	}
}

__global__ void LiftLCLKernel(const float* __restrict__ d_P, const float* __restrict__ d_T,
                              const float* __restrict__ d_PLCL, const float* __restrict__ d_Ptarget,
                              float* __restrict__ d_Tparcel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		ASSERT((d_P[idx] > 10 && d_P[idx] < 1500) || IsMissing(d_P[idx]));
		ASSERT((d_Ptarget[idx] > 10 && d_Ptarget[idx] < 1500) || IsMissing(d_Ptarget[idx]));
		ASSERT((d_T[idx] > 100 && d_T[idx] < 350) || IsMissing(d_T[idx]));

		const float T = metutil::LiftLCLA_<float>(d_P[idx] * 100, d_T[idx], d_PLCL[idx] * 100, d_Ptarget[idx] * 100);

		ASSERT((T > 100 && T < 350) || IsMissing(T));

		d_Tparcel[idx] = T;
	}
}

__global__ void MoistLiftKernel(const float* __restrict__ d_T, const float* __restrict__ d_P,
                                const float* __restrict__ d_Ptarget, float* __restrict__ d_Tparcel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		ASSERT((d_P[idx] > 10 && d_P[idx] < 1500) || IsMissing(d_P[idx]));
		ASSERT((d_Ptarget[idx] > 10 && d_Ptarget[idx] < 1500) || IsMissing(d_Ptarget[idx]));
		ASSERT((d_T[idx] > 100 && d_T[idx] < 350) || IsMissing(d_T[idx]));

		float T = metutil::MoistLiftA_<float>(d_P[idx] * 100, d_T[idx], d_Ptarget[idx] * 100);
		ASSERT((T > 100 && T < 350) || IsMissing(T));

		d_Tparcel[idx] = T;
	}
}

__global__ void CAPEKernel(const float* __restrict__ d_Tenv, const float* __restrict__ d_Penv,
                           const float* __restrict__ d_Zenv, const float* __restrict__ d_prevTenv,
                           const float* __restrict__ d_prevPenv, const float* __restrict__ d_prevZenv,
                           const float* __restrict__ d_Tparcel, const float* __restrict__ d_prevTparcel,
                           const float* __restrict__ d_LFCT, const float* __restrict__ d_LFCP,
                           float* __restrict__ d_CAPE, float* __restrict__ d_CAPE1040, float* __restrict__ d_CAPE3km,
                           float* __restrict__ d_ELT, float* __restrict__ d_ELP, float* __restrict__ d_ELZ,
                           float* __restrict__ d_LastELT, float* __restrict__ d_LastELP, float* __restrict__ d_LastELZ,
                           unsigned char* __restrict__ d_found, int d_curLevel, int d_breakLevel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N && d_found[idx] == 0)
	{
		float Tenv = d_Tenv[idx];
		ASSERT(Tenv > 100.);

		float Penv = d_Penv[idx];  // hPa
		ASSERT(Penv < 1200.);

		float Zenv = d_Zenv[idx];  // m

		float prevTenv = d_prevTenv[idx];  // K
		ASSERT(prevTenv > 100.);

		float prevPenv = d_prevPenv[idx];  // hPa
		ASSERT(prevPenv < 1200.);

		float prevZenv = d_prevZenv[idx];  // m

		float Tparcel = d_Tparcel[idx];  // K
		ASSERT(Tparcel > 100. || IsMissing(Tparcel));

		float prevTparcel = d_prevTparcel[idx];  // K
		ASSERT(prevTparcel > 100. || IsMissing(prevTparcel));

		const float LFCP = d_LFCP[idx];  // hPa
		const float LFCT = d_LFCT[idx];  // K

		if (IsMissing(Penv) || IsMissing(Tenv) || IsMissing(Zenv) || IsMissing(prevZenv) || IsMissing(Tparcel) ||
		    Penv > LFCP)
		{
			// Missing data or current grid point is below LFC
			return;
		}

		ASSERT(LFCP < 1200.);
		ASSERT(LFCT > 100.);

		if (IsMissing(prevTparcel) && !IsMissing(Tparcel))
		{
			// When rising above LFC, get accurate value of Tenv at that level so that even small amounts of CAPE
			// (and EL!) values can be determined.

			prevTenv = interpolation::Linear<float>(LFCP, prevPenv, Penv, prevTenv, Tenv);
			prevZenv = interpolation::Linear<float>(LFCP, prevPenv, Penv, prevZenv, Zenv);
			prevPenv = LFCP;     // LFC pressure
			prevTparcel = LFCT;  // LFC temperature

			// If LFC was found close to lower hybrid level, the linear interpolation and moist lift will result
			// to same values. In this case CAPE integration fails as there is no area formed between environment
			// and parcel temperature. The result for this is that LFC is found but EL is not found. To prevent
			// this, warm the parcel value just slightly so that a miniscule CAPE area is formed and EL is found.

			if (fabs(prevTparcel - prevTenv) < 0.0001)
			{
				prevTparcel += 0.0001;
			}
		}

		if (d_curLevel < d_breakLevel && (Tenv - Tparcel) > 25.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}
		else
		{
			if (prevZenv < 3000.)
			{
				float C = CAPE::CalcCAPE3km(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

				d_CAPE3km[idx] += C;

				ASSERT(d_CAPE3km[idx] >= 0);
			}

			float C = CAPE::CalcCAPE1040(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

			d_CAPE1040[idx] += C;

			ASSERT(d_CAPE1040[idx] >= 0);

			float CAPE, ELT, ELP, ELZ;
			CAPE::CalcCAPE(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv, CAPE, ELT, ELP, ELZ);

			d_CAPE[idx] += CAPE;

			ASSERT(CAPE >= 0.);

			if (IsValid(ELT))
			{
				if (IsMissing(d_ELT[idx]))
				{
					d_ELT[idx] = ELT;
				}

				d_ELP[idx] = fmaxf(ELP, d_ELP[idx]);
				d_ELZ[idx] = fminf(ELZ, d_ELZ[idx]);

				d_LastELT[idx] = ELT;
				d_LastELP[idx] = ELP;
				d_LastELZ[idx] = ELZ;
			}
		}
	}
}

__global__ void CINKernel(const float* __restrict__ d_Tenv, const float* __restrict__ d_prevTenv,
                          const float* __restrict__ d_Penv, const float* __restrict__ d_prevPenv,
                          const float* __restrict__ d_Zenv, const float* __restrict__ d_prevZenv,
                          const float* __restrict__ d_Tparcel, const float* __restrict__ d_prevTparcel,
                          const float* __restrict__ d_PLCL, const float* __restrict__ d_PLFC,
                          const float* __restrict__ d_Psource, float* __restrict__ d_cinh,
                          unsigned char* __restrict__ d_found, bool useVirtualTemperature, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N && d_found[idx] == 0)
	{
		float Tenv = d_Tenv[idx];  // K
		ASSERT(Tenv >= 150.);

		const float prevTenv = d_prevTenv[idx];

		float Penv = d_Penv[idx];  // hPa
		ASSERT(Penv < 1200. || IsMissing(Penv));

		const float prevPenv = d_prevPenv[idx];

		float Tparcel = d_Tparcel[idx];  // K
		ASSERT(Tparcel >= 150. || IsMissing(Tparcel));

		const float prevTparcel = d_prevTparcel[idx];

		const float PLFC = d_PLFC[idx];  // hPa
		ASSERT(PLFC < 1200. || IsMissing(PLFC));

		const float PLCL = d_PLCL[idx];  // hPa
		ASSERT(PLCL < 1200. || IsMissing(PLCL));

		float Zenv = d_Zenv[idx];          // m
		float prevZenv = d_prevZenv[idx];  // m

		// Make sure we have passed the starting level
		if (Penv <= d_Psource[idx])
		{
			if (Penv <= PLFC)
			{
				// reached max height
				d_found[idx] = 1;

				// Integrate the final piece from previous level to LFC level

				if (IsMissing(prevTparcel) || IsMissing(prevPenv) || IsMissing(prevTenv))
				{
					Tparcel = MissingFloat();  // unable to proceed with CIN integration
				}
				else
				{
					// First get LFC height in meters
					Zenv = interpolation::Linear<float>(PLFC, prevPenv, Penv, prevZenv, Zenv);

					// LFC environment temperature value
					Tenv = interpolation::Linear<float>(PLFC, prevPenv, Penv, prevTenv, Tenv);

					// LFC T parcel value
					Tparcel = interpolation::Linear<float>(PLFC, prevPenv, Penv, prevTparcel, Tparcel);

					Penv = PLFC;

					if (Zenv < prevZenv)
					{
						prevZenv = Zenv;
					}
				}
			}

			if (Penv < PLCL && useVirtualTemperature)
			{
				// Above LCL, switch to virtual temperature

				Tparcel = metutil::VirtualTemperature_<float>(Tparcel, Penv * 100);
				Tenv = metutil::VirtualTemperature_<float>(Tenv, Penv * 100);
			}

			if (!IsMissing(Tparcel))
			{
				d_cinh[idx] += CAPE::CalcCIN(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);
				ASSERT(d_cinh[idx] <= 0);
			}

			ASSERT(d_cinh[idx] <= 0.f);
		}
	}
}

__global__ void LFCKernel(const float* __restrict__ d_T, const float* __restrict__ d_P,
                          const float* __restrict__ d_prevT, const float* __restrict__ d_prevP,
                          float* __restrict__ d_Tparcel, const float* __restrict__ d_prevTparcel,
                          const float* __restrict__ d_LCLT, const float* __restrict__ d_LCLP,
                          float* __restrict__ d_LFCT, float* __restrict__ d_LFCP, float* __restrict__ d_LastLFCT,
                          float* __restrict__ d_LastLFCP, unsigned char* __restrict__ d_found, int d_curLevel,
                          int d_breakLevel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N && d_found[idx] == 0)
	{
		float Tparcel = d_Tparcel[idx];
		float prevTparcel = d_prevTparcel[idx];
		float Tenv = d_T[idx];

		ASSERT(Tenv < 350.);
		ASSERT(Tenv > 100.);

		float prevTenv = d_prevT[idx];
		ASSERT(prevTenv < 350.);
		ASSERT(prevTenv > 100.);

		float Penv = d_P[idx];
		float prevPenv = d_prevP[idx];

		ASSERT(Penv > 50.);
		ASSERT(Penv < 1200.);
		float LCLP = d_LCLP[idx];
		ASSERT(prevPenv > 50.);
		ASSERT(prevPenv < 1200.);

		if ((d_curLevel < d_breakLevel && (Tenv - Tparcel) > 30.) || (IsValid(d_LFCT[idx]) && Penv < 650.))
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}

		const float diff = Tparcel - Tenv;
		const float prevdiff = prevTparcel - prevTenv;
		const bool isFirstLFC = (diff >= 0 || fabs(diff) < 1e-4) && IsMissing(prevdiff) && IsMissing(d_LFCT[idx]);
		const bool isLastLFC = (diff >= 0 || fabs(diff) < 1e-4) && prevdiff < 0;

		if (d_found[idx] == 0 && Penv < LCLP && (isFirstLFC || isLastLFC))
		{
			if (IsMissing(prevTparcel))
			{
				prevTparcel = d_LCLT[idx];  // previous is LCL
				ASSERT(!IsMissing(d_LCLT[idx]));
			}

			float& Tresult = (IsMissing(d_LFCT[idx])) ? d_LFCT[idx] : d_LastLFCT[idx];
			float& Presult = (IsMissing(d_LFCP[idx])) ? d_LFCP[idx] : d_LastLFCP[idx];

			if (diff < 0.01)
			{
				Tresult = Tparcel;
				Presult = Penv;
			}
			else if (prevTparcel - prevTenv >= 0)
			{
				Tresult = prevTparcel;
				Presult = prevPenv;
			}
			else
			{
				auto intersection = CAPE::GetPointOfIntersection(point(Tenv, Penv), point(prevTenv, prevPenv),
				                                                 point(Tparcel, Penv), point(prevTparcel, prevPenv));

				Tresult = intersection.X();
				Presult = intersection.Y();

				if (Presult > prevPenv)
				{
					// Do not allow LFC to be below previous level; if intersection fails to put it in the correct
					// "bin" (between previous and current pressure), use the only information that certain:
					// the crossing has happened at least at current pressure
					Tresult = Tparcel;
					Presult = Penv;
				}
				else if (IsMissing(Tresult))
				{
					// Intersection not found, use exact level value
					Tresult = Tparcel;
					Presult = Penv;
				}
			}

			ASSERT(Tresult > 100);
			ASSERT(Tresult < 350);
		}
	}
}

__global__ void ThetaEKernel(float* __restrict__ d_T, const float* __restrict__ d_RH, float* __restrict__ d_P,
                             const float* __restrict__ d_prevT, const float* __restrict__ d_prevRH,
                             const float* __restrict__ d_prevP, float* __restrict__ d_ThetaE, float* __restrict__ d_TD,
                             unsigned char* __restrict__ d_found, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		float ThetaE = MissingFloat(), TD = MissingFloat();

		if (d_found[idx] == 0)
		{
			float& T = d_T[idx];
			float& P = d_P[idx];
			float RH = d_RH[idx];

			if (P < mucape_search_limit)
			{
				T = interpolation::Linear<float>(mucape_search_limit, P, d_prevP[idx], T, d_prevT[idx]);
				RH = interpolation::Linear<float>(mucape_search_limit, P, d_prevP[idx], RH, d_prevRH[idx]);

				d_found[idx] = 1;  // Make sure this is the last time we access this grid point
				P = mucape_search_limit;
			}

			TD = metutil::DewPointFromRH_<float>(T, RH);
			ThetaE = metutil::smarttool::ThetaE_<float>(T, RH, P * 100);
		}

		d_ThetaE[idx] = ThetaE;
		d_TD[idx] = TD;
	}
}

__global__ void MixingRatioKernel(const float* __restrict__ d_T, const float* __restrict__ d_P,
                                  const float* __restrict__ d_RH, float* __restrict__ d_Tpot, float* __restrict__ d_MR,
                                  size_t N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T);
	ASSERT(d_RH);
	ASSERT(d_P);

	if (idx < N)
	{
		const float T = d_T[idx];
		const float P = d_P[idx];
		const float RH = d_RH[idx];

		ASSERT((T > 150 && T < 350) || IsMissing(T));
		ASSERT((P > 100 && P < 1500) || IsMissing(P));
		ASSERT((RH >= 0 && RH < 102) || IsMissing(RH));

		d_Tpot[idx] = metutil::Theta_<float>(T, 100 * P);
		d_MR[idx] = metutil::smarttool::MixingRatio_<float>(T, RH, 100 * P);
	}
}

__global__ void MixingRatioFinalizeKernel(float* __restrict__ d_T, float* __restrict__ d_TD,
                                          const float* __restrict__ d_P, const float* __restrict__ d_Tpot,
                                          const float* __restrict__ d_MR, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const float P = d_P[idx];
		const float MR = d_MR[idx];
		const float Tpot = d_Tpot[idx];

		ASSERT((P > 100 && P < 1500) || IsMissing(P));

		const float T = Tpot * pow((P / 1000.), 0.2854);
		const float Es = metutil::Es_<float>(T);  // Saturated water vapor pressure
		const float E = metutil::E_<float>(MR, 100 * P);
		const float RH = fminf(102., E / Es * 100);

		d_TD[idx] = metutil::DewPointFromRH_<float>(T, RH);
		d_T[idx] = T;
	}
}

__global__ void Max1D(const float* __restrict__ d_v, unsigned char* __restrict__ d_maxima, unsigned char mask_len,
                      unsigned char K, size_t N)
{
	ASSERT(mask_len % 2 == 1);

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const unsigned char half = mask_len / 2;

		// data layout is changed here wrt to the source

		// old layout:
		// |x(0)y(0)z(0)..n(0)|x(1)y(1)z(1)..n(1)|..|x(N)y(N)z(N)..n(N)|

		// new layout:
		// |x(0)x(1)x(2)..x(K)|y(1)y(2)y(2)..y(K)|..|n(1)n(1)n(3)..n(K)|

		// beginning

		for (unsigned char i = 0; i < half; i++)
		{
			float maxv = d_v[idx];
			unsigned char maxl = 0;  // first guess

			for (unsigned char j = 1; j <= half + i; j++)
			{
				if (d_v[idx + j * N] > maxv)
				{
					maxv = d_v[idx + j * N];
					maxl = j;
				}
			}
			d_maxima[i + idx * K] = maxl;
		}

		// center

		for (unsigned char i = half; i < K - half; i++)
		{
			float maxv = d_v[idx + (i - half) * N];
			unsigned char maxl = i - half;

			for (unsigned char j = i - half + 1; j <= i + half; j++)
			{
				if (d_v[idx + j * N] > maxv)
				{
					maxv = d_v[idx + j * N];
					maxl = j;
				}
			}
			d_maxima[i + idx * K] = maxl;
		}

		// end

		for (unsigned char i = K - half; i < K; i++)
		{
			float maxv = d_v[idx + (i - half) * N];
			unsigned char maxl = i - half;

			for (unsigned char j = i - half + 1; j < K; j++)
			{
				if (d_v[idx + j * N] > maxv)
				{
					maxv = d_v[idx + j * N];
					maxl = j;
				}
			}
			d_maxima[i + idx * K] = maxl;
		}
	}
}

__global__ void MaximaLocation(const float* __restrict__ d_v, const unsigned char* __restrict__ d_maxima,
                               unsigned char* __restrict__ d_idx, size_t K, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const int maxMax = K / 4;

		int maximaN = 0;

		for (int i = 0; i < K && maximaN < (maxMax - 1); i++)
		{
			const float v = d_v[idx + i * N];  // ThetaE value at this point in the profile

			if (i == d_maxima[i + idx * K])
			{
				if (i > 0 && v == d_v[idx + (i - 1) * N])
				{
					// Duplicate maximas (two consecutive vertical levels
					// have the same thetae value and are both maximas).
					// Disregard this higher one.
				}
				else
				{
					d_idx[maximaN + 1 + idx * maxMax] = i;
					maximaN++;
				}
			}
		}

		d_idx[idx * maxMax] = maximaN;

		// bubble sort: highest value theta e should be first

		bool passed;

		do
		{
			passed = true;

			for (int i = 2; i < maximaN + 1; i++)
			{
				unsigned char& previ = d_idx[i - 1 + idx * maxMax];
				unsigned char& curi = d_idx[i + idx * maxMax];
				float prev = d_v[previ * N + idx];
				float cur = d_v[curi * N + idx];

				if (prev < cur)
				{
					unsigned char tmpi = curi;
					curi = previ;
					previ = tmpi;

					passed = false;
				}
			}
		} while (!passed);
	}
}

__global__ void MeanKernel(const float* __restrict__ d_Tpot, const float* __restrict__ d_MR,
                           const float* __restrict__ d_prevTpot, const float* __restrict__ d_prevMR,
                           float* __restrict__ d_meanTpot, float* __restrict__ d_meanMR, float* __restrict__ d_range,
                           size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		if (IsValid(d_Tpot[idx]))
		{
			// trapezoidal integration
			d_range[idx] += 1;
			d_meanTpot[idx] += (d_prevTpot[idx] + d_Tpot[idx]) * 0.5;
			d_meanMR[idx] += (d_prevMR[idx] + d_MR[idx]) * 0.5;
		}
	}
}

__global__ void MeanFinalizeKernel(float* __restrict__ d_meanTpot, float* __restrict__ d_meanMR,
                                   const float* __restrict__ d_range, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		d_meanTpot[idx] = d_meanTpot[idx] / d_range[idx];
		d_meanMR[idx] = d_meanMR[idx] / d_range[idx];
	}
}

cape_multi_source cape_cuda::GetNHighestThetaEValuesGPU(const std::shared_ptr<const plugin_configuration>& conf,
                                                        std::shared_ptr<info<float>> myTargetInfo, int n)
{
	himan::level curLevel = itsBottomLevel;

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	// We need to get the number of layers so we can preallocate
	// a suitable sized array.

	const auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), mucape_search_limit);
	const auto levelSpan = curLevel.Value() - stopLevel.second.Value();

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_T = 0;
	float* d_TD = 0;
	float* d_P = 0;
	float* d_ThetaE = 0;
	float* d_RH = 0;
	float* d_prevT = 0;
	float* d_prevP = 0;
	float* d_prevRH = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((float**)&d_T, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_TD, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_P, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ThetaE, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((float**)&d_RH, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevRH, sizeof(float) * N));

	InitializeArray<float>(d_T, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_TD, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_P, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevRH, himan::MissingFloat(), N, stream);

	InitializeArray<unsigned char>(d_found, 0, N, stream);

	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	// profiles are create as flattened vectors
	// in order the insertion to be as fast as possible, the layout is such:
	// |x(0)y(0)z(0)..n(0)|x(1)y(1)z(1)..n(1)|..|x(N)y(N)z(N)..n(N)|

	std::vector<float> ThetaEProfile(levelSpan * N), TProfile(levelSpan * N), PProfile(levelSpan * N),
	    TDProfile(levelSpan * N);

	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(ThetaEProfile.data()), sizeof(float) * levelSpan * N, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(TProfile.data()), sizeof(float) * N * levelSpan, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(TDProfile.data()), sizeof(float) * N, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(PProfile.data()), sizeof(float) * N, 0));

	const param TParam("T-K");
	const param RHParam("RH-PRCNT");
	const param PParam("P-HPA");

	size_t K = 0;  // this will hold the number of levels read (should match what we calculated previously)
	while (true)
	{
		auto TInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		auto RHInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, RHParam, myTargetInfo->ForecastType());
		auto PInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());

		if (!TInfo || !RHInfo || !PInfo)
		{
			CUDA_CHECK(hipHostUnregister(ThetaEProfile.data()));
			CUDA_CHECK(hipHostUnregister(TProfile.data()));
			CUDA_CHECK(hipHostUnregister(TDProfile.data()));
			CUDA_CHECK(hipHostUnregister(PProfile.data()));

			CUDA_CHECK(hipFree(d_T));
			CUDA_CHECK(hipFree(d_P));
			CUDA_CHECK(hipFree(d_RH));
			CUDA_CHECK(hipFree(d_prevT));
			CUDA_CHECK(hipFree(d_prevP));
			CUDA_CHECK(hipFree(d_prevRH));
			CUDA_CHECK(hipFree(d_ThetaE));
			CUDA_CHECK(hipFree(d_TD));
			CUDA_CHECK(hipFree(d_found));

			return cape_multi_source();
		}

		cuda::PrepareInfo(TInfo, d_T, stream, conf->UseCacheForReads());
		cuda::PrepareInfo(PInfo, d_P, stream, conf->UseCacheForReads());
		cuda::PrepareInfo(RHInfo, d_RH, stream, conf->UseCacheForReads());

		ThetaEKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_RH, d_P, d_prevT, d_prevRH, d_prevP, d_ThetaE, d_TD,
		                                                 d_found, N);

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);

		CUDA_CHECK(hipMemcpyAsync(&ThetaEProfile[K * N], d_ThetaE, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&TProfile[K * N], d_T, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&TDProfile[K * N], d_TD, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&PProfile[K * N], d_P, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));

		curLevel.Value(curLevel.Value() - 1);
		K++;

		if (foundCount == N || levelSpan == K)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevT, d_T, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevP, d_P, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevRH, d_RH, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
	}

	CUDA_CHECK(hipHostUnregister(ThetaEProfile.data()));
	CUDA_CHECK(hipHostUnregister(TProfile.data()));
	CUDA_CHECK(hipHostUnregister(TDProfile.data()));
	CUDA_CHECK(hipHostUnregister(PProfile.data()));

	CUDA_CHECK(hipFree(d_T));
	CUDA_CHECK(hipFree(d_P));
	CUDA_CHECK(hipFree(d_RH));
	CUDA_CHECK(hipFree(d_prevT));
	CUDA_CHECK(hipFree(d_prevP));
	CUDA_CHECK(hipFree(d_prevRH));
	CUDA_CHECK(hipFree(d_ThetaE));
	CUDA_CHECK(hipFree(d_TD));
	CUDA_CHECK(hipFree(d_found));

	// Check comments from cape.cpp
	vec2d Tret(n), TDret(n), Pret(n);

	for (size_t j = 0; j < static_cast<size_t>(n); j++)
	{
		Tret[j].resize(N, MissingFloat());
		TDret[j].resize(N, MissingFloat());
		Pret[j].resize(N, MissingFloat());
	}

	float* d_v = 0;
	unsigned char* d_maxima = 0;
	unsigned char* d_idxs = 0;

	CUDA_CHECK(hipMalloc((float**)&d_v, sizeof(float) * N * K));  // Actual ThetaE values
	CUDA_CHECK(hipMalloc((unsigned char**)&d_maxima,
	                      sizeof(unsigned char) * N * K));  // Local maxima locations in the profile

	CUDA_CHECK(hipMemcpyAsync(d_v, ThetaEProfile.data(), sizeof(float) * N * K, hipMemcpyHostToDevice, stream));

	Max1D<<<gridSize, blockSize, 0, stream>>>(d_v, d_maxima, 5, K, N);

	// maximum number of maximas we expect to find in the profile
	const size_t maxMax = K / 4;

	CUDA_CHECK(hipMalloc((unsigned char**)&d_idxs, sizeof(unsigned char) * N * maxMax));

	MaximaLocation<<<gridSize, blockSize, 0, stream>>>(d_v, d_maxima, d_idxs, K, N);

	std::vector<unsigned char> idxs(N * maxMax);

	CUDA_CHECK(hipMemcpyAsync(&idxs[0], d_idxs, sizeof(unsigned char) * N * maxMax, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_v));
	CUDA_CHECK(hipFree(d_maxima));
	CUDA_CHECK(hipFree(d_idxs));

	for (size_t i = 0; i < N; i++)
	{
		const size_t s = i * maxMax;     // start index of this grid point
		const size_t maximaN = idxs[s];  // number of maximas found (at most maxMax)

		ASSERT(maximaN > 0);
		ASSERT(maximaN <= maxMax);

		// Remove maximas that are too high in the atmosphere
		size_t newMaximaN = 0;
		size_t offset = 0;
		for (size_t j = 0; j < maximaN; j++)
		{
			const size_t sidx = 1 + s + j;            // index in the array where maxima index is found
			const unsigned char maxidx = idxs[sidx];  // index in the vertical profile where the maxima was found

			if (PProfile[maxidx * N + i] < mucape_maxima_search_limit)
			{
				offset++;
				continue;
			}
			newMaximaN++;
			idxs[sidx - offset] = maxidx;
		}

		for (size_t j = 0; j < min(static_cast<size_t>(n), newMaximaN); j++)
		{
			const size_t sidx = 1 + s + j;  // index in the array where maxima index is found
			const short maxidx =
			    static_cast<short>(idxs[sidx]);  // index in the vertical profile where the maxima was found

			ASSERT(static_cast<unsigned>(maxidx) <= K);

			// Copy values from max theta e levels for further processing

			Tret[j][i] = TProfile[maxidx * N + i];
			TDret[j][i] = TDProfile[maxidx * N + i];
			Pret[j][i] = PProfile[maxidx * N + i];

			ASSERT(IsValid(Tret[j][i]));
			ASSERT(IsValid(TDret[j][i]));
			ASSERT(IsValid(Pret[j][i]));
		}
	}

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_tuple(Tret, TDret, Pret);
}

cape_source cape_cuda::Get500mMixingRatioValuesGPU(std::shared_ptr<const plugin_configuration>& conf,
                                                   std::shared_ptr<info<float>> myTargetInfo)
{
	myTargetInfo->FirstValidGrid();
	const size_t N = myTargetInfo->Data().Size();

	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	level curLevel = itsBottomLevel;

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());

	modifier_mean tp, mr;

	tp.HeightInMeters(false);
	mr.HeightInMeters(false);

	auto PInfo = cuda::Fetch<double>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType(), false);

	if (!PInfo || PInfo->Data().MissingCount() == PInfo->SizeLocations())
	{
		return std::make_tuple(std::vector<float>(), std::vector<float>(), std::vector<float>());
	}

	auto dPVec = VEC(PInfo);

	auto P500m = h->VerticalValue<double>(PParam, 500.);
	auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), 500.);

	h->HeightUnit(kHPa);

	tp.LowerHeight(dPVec);
	mr.LowerHeight(dPVec);

	tp.UpperHeight(P500m);
	mr.UpperHeight(P500m);

	auto PVec = util::Convert<double, float>(dPVec);

	auto sourceData = GetSampledSourceData(conf, myTargetInfo, util::Convert<double, float>(P500m), PVec,
	                                       itsBottomLevel, stopLevel.second);

	float* d_Tpot = 0;
	float* d_MR = 0;
	float* d_prevTpot = 0;
	float* d_prevMR = 0;
	float* d_meanTpot = 0;
	float* d_meanMR = 0;
	float* d_range = 0;
	float* d_T = 0;
	float* d_RH = 0;
	float* d_P = 0;
	float* d_TD = 0;

	CUDA_CHECK(hipMalloc((float**)&d_Tpot, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_MR, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevTpot, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevMR, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_meanTpot, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_meanMR, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_range, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_T, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_RH, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_P, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_TD, N * sizeof(float)));

	InitializeArray<float>(d_meanTpot, 0.f, N, stream);
	InitializeArray<float>(d_meanMR, 0.f, N, stream);
	InitializeArray<float>(d_range, 0.f, N, stream);

	const auto& Psample = std::get<0>(sourceData);
	const auto& Tsample = std::get<1>(sourceData);
	const auto& RHsample = std::get<2>(sourceData);

	std::vector<float> T(N, MissingFloat());
	std::vector<float> RH(N, MissingFloat());
	std::vector<float> P(N, MissingFloat());

	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(T.data()), sizeof(float) * N, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(P.data()), sizeof(float) * N, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(RH.data()), sizeof(float) * N, 0));

	unsigned int k = 0;

	while (true)
	{
		CUDA_CHECK(hipStreamSynchronize(stream));

		for (size_t i = 0; i < N; i++)
		{
			if (k >= Psample[i].size())
			{
				T[i] = RH[i] = P[i] = MissingFloat();
				continue;
			}

			T[i] = Tsample[i][k];
			RH[i] = RHsample[i][k];
			P[i] = Psample[i][k];
		}

		if (static_cast<unsigned int>(count_if(P.begin(), P.end(), [](const float& v) { return IsMissing(v); })) ==
		    P.size())
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_T, T.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_RH, RH.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_P, P.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));

		MixingRatioKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_P, d_RH, d_Tpot, d_MR, N);

		if (k == 0)
		{
			CUDA_CHECK(hipMemcpyAsync(d_prevTpot, d_Tpot, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
			CUDA_CHECK(hipMemcpyAsync(d_prevMR, d_MR, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

			k++;
			continue;
		}

		MeanKernel<<<gridSize, blockSize, 0, stream>>>(d_Tpot, d_MR, d_prevTpot, d_prevMR, d_meanTpot, d_meanMR,
		                                               d_range, N);

		CUDA_CHECK(hipMemcpyAsync(d_prevTpot, d_Tpot, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevMR, d_MR, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		k++;
	}

	CUDA_CHECK(hipHostUnregister(T.data()));
	CUDA_CHECK(hipHostUnregister(P.data()));
	CUDA_CHECK(hipHostUnregister(RH.data()));

	MeanFinalizeKernel<<<gridSize, blockSize, 0, stream>>>(d_meanTpot, d_meanMR, d_range, N);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_Tpot));
	CUDA_CHECK(hipFree(d_MR));
	CUDA_CHECK(hipFree(d_prevTpot));
	CUDA_CHECK(hipFree(d_prevMR));
	CUDA_CHECK(hipFree(d_range));

	float* d_Psurf = 0;
	CUDA_CHECK(hipMalloc((float**)&d_Psurf, N * sizeof(float)));

	auto Psurf = cuda::Fetch<float>(conf, myTargetInfo->Time(), itsBottomLevel, PParam, myTargetInfo->ForecastType());
	cuda::PrepareInfo(Psurf, d_Psurf, stream, conf->UseCacheForReads());

	std::vector<float> TD(T.size());

	MixingRatioFinalizeKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_TD, d_Psurf, d_meanTpot, d_meanMR, N);

	CUDA_CHECK(hipMemcpyAsync(T.data(), d_T, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(TD.data(), d_TD, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_meanTpot));
	CUDA_CHECK(hipFree(d_meanMR));
	CUDA_CHECK(hipFree(d_RH));
	CUDA_CHECK(hipFree(d_P));
	CUDA_CHECK(hipFree(d_T));
	CUDA_CHECK(hipFree(d_TD));
	CUDA_CHECK(hipFree(d_Psurf));

	CUDA_CHECK(hipStreamDestroy(stream));
	return std::make_tuple(T, TD, VEC(Psurf));
}

std::vector<std::pair<std::vector<float>, std::vector<float>>> cape_cuda::GetLFCGPU(
    const std::shared_ptr<const plugin_configuration>& conf, std::shared_ptr<info<float>> myTargetInfo,
    std::vector<float>& T, std::vector<float>& P, std::vector<float>& TenvLCL)
{
	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_LCLP = 0;
	float* d_LCLT = 0;
	float* d_LFCT = 0;
	float* d_LFCP = 0;
	float* d_LastLFCT = 0;
	float* d_LastLFCP = 0;
	float* d_Tparcel = 0;
	float* d_prevTparcel = 0;
	float* d_Tenv = 0;
	float* d_Penv = 0;
	float* d_prevTenv = 0;
	float* d_prevPenv = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((float**)&d_LCLT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LCLP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LFCT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LFCP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastLFCT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastLFCP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Penv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevPenv, sizeof(float) * N));

	CUDA_CHECK(hipMemcpyAsync(d_LCLT, &T[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LCLP, &P[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));

	InitializeArray<float>(d_LFCT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LFCP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastLFCT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastLFCP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevTparcel, himan::MissingFloat(), N, stream);
	InitializeArray<unsigned char>(d_found, 0, N, stream);

	// For each grid point find the hybrid level that's below LCL and then pick the lowest level
	// among all grid points; most commonly it's the lowest hybrid level

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P));

	level curLevel = levels.first;

	auto prevPenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());
	auto prevTenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());

	cuda::PrepareInfo(prevTenvInfo, d_prevTenv, stream, conf->UseCacheForReads());
	cuda::PrepareInfo(prevPenvInfo, d_prevPenv, stream, conf->UseCacheForReads());

	if (cape_cuda::itsUseVirtualTemperature)
	{
		VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_prevTenv, d_prevPenv, N);
	}

	curLevel.Value(curLevel.Value() - 1);

	std::vector<unsigned char> found(N, 0);
	std::vector<float> LFCT(N, himan::MissingFloat());
	std::vector<float> LFCP(N, himan::MissingFloat());
	std::vector<float> LastLFCT(N);
	std::vector<float> LastLFCP(N);

	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	for (size_t i = 0; i < N; i++)
	{
		if ((T[i] - TenvLCL[i]) > 0.0001)
		{
			LFCT[i] = T[i];
			LFCP[i] = P[i];
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCT, &LFCT[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCP, &LFCP[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	auto hPa450 = h->LevelForHeight(myTargetInfo->Producer(), 450.);
	auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), 250.);

	while (curLevel.Value() > stopLevel.first.Value())
	{
		// Get environment temperature and pressure values for this level
		auto TenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		auto PenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());

		cuda::PrepareInfo(PenvInfo, d_Penv, stream, conf->UseCacheForReads());
		cuda::PrepareInfo(TenvInfo, d_Tenv, stream, conf->UseCacheForReads());

		if (cape_cuda::itsUseVirtualTemperature)
		{
			VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, N);
		}

		// Lift the particle from previous level to this level. In the first revolution
		// of this loop the starting level is LCL. If target level level is below current level
		// (ie. we would be lowering the particle) missing value is returned.

		MoistLiftKernel<<<gridSize, blockSize, 0, stream>>>(d_LCLT, d_LCLP, d_Penv, d_Tparcel, N);

		LFCKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, d_prevTenv, d_prevPenv, d_Tparcel, d_prevTparcel,
		                                              d_LCLT, d_LCLP, d_LFCT, d_LFCP, d_LastLFCT, d_LastLFCP, d_found,
		                                              curLevel.Value(), hPa450.first.Value(), N);

		CUDA_CHECK(hipStreamSynchronize(stream));

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (N == foundCount)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTenv, d_Tenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevPenv, d_Penv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		curLevel.Value(curLevel.Value() - 1);
	}

	LastLFCCopyKernel<<<gridSize, blockSize, 0, stream>>>(d_LFCT, d_LFCP, d_LastLFCT, d_LastLFCP, N);

	CUDA_CHECK(hipMemcpyAsync(LFCT.data(), d_LFCT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(LFCP.data(), d_LFCP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(LastLFCT.data(), d_LastLFCT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(LastLFCP.data(), d_LastLFCP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipFree(d_LCLT));
	CUDA_CHECK(hipFree(d_LCLP));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_Penv));
	CUDA_CHECK(hipFree(d_Tenv));
	CUDA_CHECK(hipFree(d_prevPenv));
	CUDA_CHECK(hipFree(d_prevTenv));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_LastLFCT));
	CUDA_CHECK(hipFree(d_LastLFCP));

	CUDA_CHECK(hipStreamDestroy(stream));

	return {std::make_pair(LFCT, LFCP), std::make_pair(LastLFCT, LastLFCP)};
}

std::vector<float> cape_cuda::GetCINGPU(const std::shared_ptr<const plugin_configuration>& conf,
                                        std::shared_ptr<info<float>> myTargetInfo, const std::vector<float>& Tsource,
                                        const std::vector<float>& Psource, const std::vector<float>& PLCL,
                                        const std::vector<float>& PLFC, const std::vector<float>& ZLFC)
{
	const params PParams({param("PGR-PA"), param("P-PA")});

	forecast_time ftime = myTargetInfo->Time();
	forecast_type ftype = myTargetInfo->ForecastType();

	/*
	 * Modus operandi:
	 *
	 * 1. Integrate from ground to LCL dry adiabatically
	 *
	 * This can be done always since LCL is known at all grid points
	 * (that have source data values defined).
	 *
	 * 2. Integrate from LCL to LFC moist adiabatically
	 *
	 * Note! For some points integration will fail (no LFC found)
	 *
	 * We stop integrating at first time CAPE area is found!
	 */

	level curLevel = itsBottomLevel;

	auto prevZenvInfo = cuda::Fetch<float>(conf, ftime, curLevel, ZParam, ftype);
	auto prevTenvInfo = cuda::Fetch<float>(conf, ftime, curLevel, TParam, ftype);
	auto prevPenvInfo = cuda::Fetch<float>(conf, ftime, curLevel, PParam, ftype);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_Psource = 0;
	float* d_Tparcel = 0;
	float* d_prevTparcel = 0;
	float* d_Tsource = 0;
	float* d_PLCL = 0;
	float* d_PLFC = 0;
	float* d_cinh = 0;
	float* d_prevZenv = 0;
	float* d_prevTenv = 0;
	float* d_prevPenv = 0;
	float* d_Zenv = 0;
	float* d_Tenv = 0;
	float* d_Penv = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((float**)&d_Psource, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tparcel, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevTparcel, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tsource, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_PLCL, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_PLFC, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_cinh, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevZenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevTenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevPenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Zenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Penv, N * sizeof(float)));

	CUDA_CHECK(hipMalloc((unsigned char**)&d_found, N * sizeof(unsigned char)));

	cuda::PrepareInfo(prevZenvInfo, d_prevZenv, stream, conf->UseCacheForReads());
	cuda::PrepareInfo(prevTenvInfo, d_prevTenv, stream, conf->UseCacheForReads());
	cuda::PrepareInfo(prevPenvInfo, d_prevPenv, stream, conf->UseCacheForReads());

	InitializeArray<float>(d_cinh, 0., N, stream);
	InitializeArray<float>(d_Tparcel, himan::MissingFloat(), N, stream);

	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, Tsource.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Psource, Psource.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Tsource, d_prevTparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLCL, PLCL.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLFC, PLFC.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));

	std::vector<unsigned char> found(N, 0);

	for (size_t i = 0; i < PLFC.size(); i++)
	{
		if (IsMissing(PLFC[i]))
		{
			found[i] = true;
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));

	curLevel.Value(curLevel.Value() - 1);

	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	auto hPa100 = h->LevelForHeight(myTargetInfo->Producer(), 100.);
	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	while (curLevel.Value() > hPa100.first.Value())
	{
		auto ZenvInfo = cuda::Fetch<float>(conf, ftime, curLevel, ZParam, ftype);
		auto TenvInfo = cuda::Fetch<float>(conf, ftime, curLevel, TParam, ftype);
		auto PenvInfo = cuda::Fetch<float>(conf, ftime, curLevel, PParam, ftype);

		cuda::PrepareInfo(ZenvInfo, d_Zenv, stream, conf->UseCacheForReads());
		cuda::PrepareInfo(PenvInfo, d_Penv, stream, conf->UseCacheForReads());
		cuda::PrepareInfo(TenvInfo, d_Tenv, stream, conf->UseCacheForReads());

		LiftLCLKernel<<<gridSize, blockSize, 0, stream>>>(d_Psource, d_Tsource, d_PLCL, d_Penv, d_Tparcel, N);

		CINKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_prevTenv, d_Penv, d_prevPenv, d_Zenv, d_prevZenv,
		                                              d_Tparcel, d_prevTparcel, d_PLCL, d_PLFC, d_Psource, d_cinh,
		                                              d_found, cape_cuda::itsUseVirtualTemperature, N);

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);
		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (N == foundCount)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevZenv, d_Zenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTenv, d_Tenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevPenv, d_Penv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		curLevel.Value(curLevel.Value() - 1);
	}

	std::vector<float> cinh(N, 0);

	CUDA_CHECK(hipMemcpyAsync(&cinh[0], d_cinh, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipFree(d_Psource));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_Tsource));
	CUDA_CHECK(hipFree(d_PLCL));
	CUDA_CHECK(hipFree(d_PLFC));
	CUDA_CHECK(hipFree(d_prevZenv));
	CUDA_CHECK(hipFree(d_prevPenv));
	CUDA_CHECK(hipFree(d_prevTenv));
	CUDA_CHECK(hipFree(d_Zenv));
	CUDA_CHECK(hipFree(d_Penv));
	CUDA_CHECK(hipFree(d_Tenv));
	CUDA_CHECK(hipFree(d_found));

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_cinh));

	CUDA_CHECK(hipStreamDestroy(stream));

	return cinh;
}

CAPEdata cape_cuda::GetCAPEGPU(const std::shared_ptr<const plugin_configuration>& conf,
                               std::shared_ptr<info<float>> myTargetInfo, const std::vector<float>& T,
                               const std::vector<float>& P)
{
	ASSERT(T.size() == P.size());

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	// Typically LFC value is not found to all grid points -- in many cases as few as
	// 10% of grid points have LFC defined. Therefore we do some memory-optimization
	// here: instead of allocating and processing whole grids with 90% missing data,
	// we 'compact' the data with a bitmap.
	//
	// The LFC pressure field is read and each missing grid point is marked with a '1'.
	// The effective grid size in the processing phase is then the number of ones in the
	// bitmap field. Every time a a new field is read from database, it is processed so
	// that grid points who's bitmap value is zero are removed.
	//
	// At the end of processing the grid is again 'expanded' so that the grid point values
	// in the compact grid are moved to correct locations.

	std::vector<unsigned char> bitmap(T.size(), 1);

	// No LFC --> No CAPE

	for (size_t i = 0; i < P.size(); i++)
	{
		if (IsMissing(P[i]))
		{
			bitmap[i] = 0;
		}
	}

	const size_t N = count(bitmap.begin(), bitmap.end(), 1);
	const size_t NB = bitmap.size();

	// 'bitmap' variable is immutable; 'found' variable gets filled up with
	// ones as processing proceeds

	std::vector<unsigned char> found(N, 0);

	logger log("CAPEGPU");

	log.Info("Compacting data with a ratio of " + std::to_string(static_cast<double>(N) / NB) + " (smaller is better)");

	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_CAPE = 0;
	float* d_CAPE1040 = 0;
	float* d_CAPE3km = 0;
	float* d_ELT = 0;
	float* d_ELP = 0;
	float* d_ELZ = 0;
	float* d_LastELT = 0;
	float* d_LastELP = 0;
	float* d_LastELZ = 0;
	float* d_prevTparcel = 0;
	float* d_Tparcel = 0;
	float* d_LFCT = 0;
	float* d_LFCP = 0;
	float* d_origLFCT = 0;
	float* d_origLFCP = 0;
	float* d_prevZenv = 0;
	float* d_prevPenv = 0;
	float* d_prevTenv = 0;
	float* d_Zenv = 0;
	float* d_Penv = 0;
	float* d_Tenv = 0;
	float* d_origZenv = 0;
	float* d_origPenv = 0;
	float* d_origTenv = 0;

	unsigned char* d_found = 0;
	unsigned char* d_bitmap = 0;

	CUDA_CHECK(hipMalloc((float**)&d_CAPE, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_CAPE1040, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_CAPE3km, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ELP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ELT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ELZ, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastELP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastELT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastELZ, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevZenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevPenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Zenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Penv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_origZenv, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_origTenv, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_origPenv, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_LFCT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LFCP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_origLFCT, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_origLFCP, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((unsigned char**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((unsigned char**)&d_bitmap, sizeof(unsigned char) * NB));

	CUDA_CHECK(hipMemcpyAsync(d_origLFCP, P.data(), sizeof(float) * NB, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_origLFCT, T.data(), sizeof(float) * NB, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_bitmap, bitmap.data(), sizeof(unsigned char) * NB, hipMemcpyHostToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_LFCT, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_bitmap, bitmap.data(), sizeof(unsigned char) * NB, hipMemcpyHostToDevice, stream));

	auto bitmapHot = [] __device__(const unsigned char& u) { return u == 1; };

	// https://thrust.github.io/doc/group__stream__compaction.html#ga36d9d6ed8e17b442c1fd8dc40bd515d5
	thrust::copy_if(thrust::cuda::par.on(stream), d_origLFCT, d_origLFCT + NB, d_bitmap, d_LFCT, bitmapHot);
	thrust::copy_if(thrust::cuda::par.on(stream), d_origLFCP, d_origLFCP + NB, d_bitmap, d_LFCP, bitmapHot);

	InitializeArray<unsigned char>(d_found, 0., N, stream);
	InitializeArray<float>(d_CAPE, 0., N, stream);
	InitializeArray<float>(d_CAPE1040, 0., N, stream);
	InitializeArray<float>(d_CAPE3km, 0., N, stream);

	InitializeArray<float>(d_ELP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_ELT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_ELZ, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastELP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastELT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastELZ, himan::MissingFloat(), N, stream);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_origLFCT));
	CUDA_CHECK(hipFree(d_origLFCP));

	// For each grid point find the hybrid level that's below LFC and then pick the lowest level
	// among all grid points

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P));

	level curLevel = levels.first;

	auto prevZenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, ZParam, myTargetInfo->ForecastType());
	auto prevTenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
	auto prevPenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());

	// "orig" variables are just as a temporary placeholder
	cuda::PrepareInfo(prevZenvInfo, d_origZenv, stream, conf->UseCacheForReads());
	cuda::PrepareInfo(prevPenvInfo, d_origPenv, stream, conf->UseCacheForReads());
	cuda::PrepareInfo(prevTenvInfo, d_origTenv, stream, conf->UseCacheForReads());

	thrust::copy_if(thrust::cuda::par.on(stream), d_origZenv, d_origZenv + NB, d_bitmap, d_prevZenv, bitmapHot);
	thrust::copy_if(thrust::cuda::par.on(stream), d_origPenv, d_origPenv + NB, d_bitmap, d_prevPenv, bitmapHot);
	thrust::copy_if(thrust::cuda::par.on(stream), d_origTenv, d_origTenv + NB, d_bitmap, d_prevTenv, bitmapHot);

	if (cape_cuda::itsUseVirtualTemperature)
	{
		VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_prevTenv, d_prevPenv, N);
	}

	curLevel.Value(curLevel.Value());

	auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), 50.);
	auto hPa450 = h->LevelForHeight(myTargetInfo->Producer(), 450.);

	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	std::shared_ptr<info<float>> PenvInfo, TenvInfo, ZenvInfo;

	while (curLevel.Value() > stopLevel.first.Value())
	{
		PenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());
		TenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		ZenvInfo = cuda::Fetch<float>(conf, myTargetInfo->Time(), curLevel, ZParam, myTargetInfo->ForecastType());

		if (!PenvInfo || !TenvInfo || !ZenvInfo)
		{
			break;
		}

		cuda::PrepareInfo<float>(ZenvInfo, d_origZenv, stream, conf->UseCacheForReads());
		cuda::PrepareInfo<float>(PenvInfo, d_origPenv, stream, conf->UseCacheForReads());
		cuda::PrepareInfo<float>(TenvInfo, d_origTenv, stream, conf->UseCacheForReads());

		thrust::copy_if(thrust::cuda::par.on(stream), d_origZenv, d_origZenv + NB, d_bitmap, d_Zenv, bitmapHot);
		thrust::copy_if(thrust::cuda::par.on(stream), d_origPenv, d_origPenv + NB, d_bitmap, d_Penv, bitmapHot);
		thrust::copy_if(thrust::cuda::par.on(stream), d_origTenv, d_origTenv + NB, d_bitmap, d_Tenv, bitmapHot);

		if (cape_cuda::itsUseVirtualTemperature)
		{
			VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, N);
		}

		MoistLiftKernel<<<gridSize, blockSize, 0, stream>>>(d_LFCT, d_LFCP, d_Penv, d_Tparcel, N);

		CAPEKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, d_Zenv, d_prevTenv, d_prevPenv, d_prevZenv,
		                                               d_Tparcel, d_prevTparcel, d_LFCT, d_LFCP, d_CAPE, d_CAPE1040,
		                                               d_CAPE3km, d_ELT, d_ELP, d_ELZ, d_LastELT, d_LastELP, d_LastELZ,
		                                               d_found, curLevel.Value(), hPa450.first.Value(), N);

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);
		CUDA_CHECK(hipStreamSynchronize(stream));

		if (foundCount == N)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevZenv, d_Zenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTenv, d_Tenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevPenv, d_Penv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		curLevel.Value(curLevel.Value() - 1);
	}

	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_bitmap));
	CUDA_CHECK(hipFree(d_prevTenv));
	CUDA_CHECK(hipFree(d_prevPenv));
	CUDA_CHECK(hipFree(d_prevZenv));
	CUDA_CHECK(hipFree(d_origTenv));
	CUDA_CHECK(hipFree(d_origPenv));
	CUDA_CHECK(hipFree(d_origZenv));

	CapELValuesKernel<<<gridSize, blockSize, 0, stream>>>(d_CAPE, d_ELT, d_ELP, d_ELZ, d_LastELT, d_LastELP, d_LastELZ,
	                                                      d_Tenv, d_Penv, d_Zenv, N);

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_Tenv));
	CUDA_CHECK(hipFree(d_Penv));
	CUDA_CHECK(hipFree(d_Zenv));

	std::vector<float> CAPE(NB, 0);
	std::vector<float> CAPE1040(NB, 0);
	std::vector<float> CAPE3km(NB, 0);
	std::vector<float> ELT(NB, himan::MissingFloat());
	std::vector<float> ELP(NB, himan::MissingFloat());
	std::vector<float> ELZ(NB, himan::MissingFloat());
	std::vector<float> LastELT(NB, himan::MissingFloat());
	std::vector<float> LastELP(NB, himan::MissingFloat());
	std::vector<float> LastELZ(NB, himan::MissingFloat());

	// intermediate containers where data is copied before expansion
	std::vector<float> bm_CAPE(N);
	std::vector<float> bm_CAPE1040(N);
	std::vector<float> bm_CAPE3km(N);
	std::vector<float> bm_ELT(N);
	std::vector<float> bm_ELP(N);
	std::vector<float> bm_ELZ(N);
	std::vector<float> bm_LastELT(N);
	std::vector<float> bm_LastELP(N);
	std::vector<float> bm_LastELZ(N);

	CUDA_CHECK(hipMemcpyAsync(bm_CAPE.data(), d_CAPE, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_CAPE1040.data(), d_CAPE1040, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_CAPE3km.data(), d_CAPE3km, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_ELT.data(), d_ELT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_ELP.data(), d_ELP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_ELZ.data(), d_ELZ, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_LastELT.data(), d_LastELT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_LastELP.data(), d_LastELP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_LastELZ.data(), d_LastELZ, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_CAPE));
	CUDA_CHECK(hipFree(d_CAPE1040));
	CUDA_CHECK(hipFree(d_CAPE3km));
	CUDA_CHECK(hipFree(d_ELT));
	CUDA_CHECK(hipFree(d_ELP));
	CUDA_CHECK(hipFree(d_ELZ));
	CUDA_CHECK(hipFree(d_LastELT));
	CUDA_CHECK(hipFree(d_LastELP));
	CUDA_CHECK(hipFree(d_LastELZ));

	// "expand" data again with bitmap

	for (size_t i = 0, j = 0; i < NB; i++)
	{
		if (bitmap[i])
		{
			CAPE[i] = bm_CAPE[j];
			CAPE1040[i] = bm_CAPE1040[j];
			CAPE3km[i] = bm_CAPE3km[j];
			ELT[i] = bm_ELT[j];
			ELP[i] = bm_ELP[j];
			ELZ[i] = bm_ELZ[j];
			LastELT[i] = bm_LastELT[j];
			LastELP[i] = bm_LastELP[j];
			LastELZ[i] = bm_LastELZ[j];

			j++;
		}
	}

	return make_tuple(ELT, ELP, ELZ, LastELT, LastELP, LastELZ, CAPE, CAPE1040, CAPE3km);
}
