#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/execution_policy.h>

#include "plugin_factory.h"

#include "cape.cuh"
#include "cuda_helper.h"
#include "lift.h"
#include "util.h"

#include "cuda_plugin_helper.h"
#include "forecast_time.h"
#include "level.h"
#include "timer.h"

#define HIMAN_AUXILIARY_INCLUDE

#include "hitool.h"

#include "debug.h"

#undef HIMAN_AUXILIARY_INCLUDE

using namespace himan;
using namespace himan::numerical_functions;
using namespace himan::plugin;
namespace hc = himan::cuda;

himan::level cape_cuda::itsBottomLevel;
bool cape_cuda::itsUseVirtualTemperature;

typedef std::vector<std::vector<float>> vec2d;

extern float Max(const std::vector<float>& vec);

template <typename T>
__global__ void InitializeArrayKernel(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = val;
	}
}

template <typename T>
void InitializeArray(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	InitializeArrayKernel<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

template <typename T>
__global__ void MultiplyWith(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = d_arr[idx] * val;
	}
}

template <typename T>
void MultiplyWith(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	MultiplyWith<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

__global__ void LastLFCCopyKernel(const float* __restrict__ d_LFCT, const float* __restrict__ d_LFCP,
                                  float* __restrict__ d_LastLFCT, float* __restrict__ d_LastLFCP, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		if (IsMissing(d_LastLFCT[idx]) || d_LastLFCP[idx] > d_LFCP[idx])
		{
			d_LastLFCT[idx] = d_LFCT[idx];
			d_LastLFCP[idx] = d_LFCP[idx];
		}
	}
}

__global__ void CapELValuesKernel(const float* __restrict__ d_CAPE, float* __restrict__ d_ELT,
                                  float* __restrict__ d_ELP, float* __restrict__ d_ELZ, float* __restrict__ d_LastELT,
                                  float* __restrict__ d_LastELP, float* __restrict__ d_LastELZ,
                                  const float* __restrict__ d_Tenv, const float* __restrict__ d_Penv,
                                  const float* __restrict__ d_Zenv, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		// If the CAPE area is continued all the way to stopLevel and beyond, we don't have an EL for that
		// (since integration is forcefully stopped)
		// In this case let last level be EL

		if (d_CAPE[idx] > 0 && IsMissing(d_ELT[idx]))
		{
			d_ELT[idx] = d_Tenv[idx];
			d_ELP[idx] = d_Penv[idx];
			d_ELZ[idx] = d_Zenv[idx];

			d_LastELT[idx] = d_Tenv[idx];
			d_LastELP[idx] = d_Penv[idx];
			d_LastELZ[idx] = d_Zenv[idx];
		}
	}
}

__global__ void VirtualTemperatureKernel(float* __restrict__ d_T, const float* __restrict__ d_P, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		d_T[idx] = himan::metutil::VirtualTemperature_<float>(d_T[idx], d_P[idx] * 100);
	}
}

__global__ void LiftLCLKernel(const float* __restrict__ d_P, const float* __restrict__ d_T,
                              const float* __restrict__ d_PLCL, const float* __restrict__ d_Ptarget,
                              float* __restrict__ d_Tparcel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		ASSERT((d_P[idx] > 10 && d_P[idx] < 1500) || IsMissing(d_P[idx]));
		ASSERT((d_Ptarget[idx] > 10 && d_Ptarget[idx] < 1500) || IsMissing(d_Ptarget[idx]));
		ASSERT((d_T[idx] > 100 && d_T[idx] < 350) || IsMissing(d_T[idx]));

		const float T = metutil::LiftLCLA_<float>(d_P[idx] * 100, d_T[idx], d_PLCL[idx] * 100, d_Ptarget[idx] * 100);

		ASSERT((T > 100 && T < 350) || IsMissing(T));

		d_Tparcel[idx] = T;
	}
}

__global__ void MoistLiftKernel(const float* __restrict__ d_T, const float* __restrict__ d_P,
                                const float* __restrict__ d_Ptarget, float* __restrict__ d_Tparcel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		ASSERT((d_P[idx] > 10 && d_P[idx] < 1500) || IsMissing(d_P[idx]));
		ASSERT((d_Ptarget[idx] > 10 && d_Ptarget[idx] < 1500) || IsMissing(d_Ptarget[idx]));
		ASSERT((d_T[idx] > 100 && d_T[idx] < 350) || IsMissing(d_T[idx]));

		float T = metutil::MoistLiftA_<float>(d_P[idx] * 100, d_T[idx], d_Ptarget[idx] * 100);
		ASSERT((T > 100 && T < 350) || IsMissing(T));

		d_Tparcel[idx] = T;
	}
}

__global__ void CAPEKernel(const float* __restrict__ d_Tenv, const float* __restrict__ d_Penv,
                           const float* __restrict__ d_Zenv, const float* __restrict__ d_prevTenv,
                           const float* __restrict__ d_prevPenv, const float* __restrict__ d_prevZenv,
                           const float* __restrict__ d_Tparcel, const float* __restrict__ d_prevTparcel,
                           const float* __restrict__ d_LFCT, const float* __restrict__ d_LFCP,
                           float* __restrict__ d_CAPE, float* __restrict__ d_CAPE1040, float* __restrict__ d_CAPE3km,
                           float* __restrict__ d_ELT, float* __restrict__ d_ELP, float* __restrict__ d_ELZ,
                           float* __restrict__ d_LastELT, float* __restrict__ d_LastELP, float* __restrict__ d_LastELZ,
                           unsigned char* __restrict__ d_found, int d_curLevel, int d_breakLevel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N && d_found[idx] == 0)
	{
		float Tenv = d_Tenv[idx];
		ASSERT(Tenv > 100.);

		float Penv = d_Penv[idx];  // hPa
		ASSERT(Penv < 1200.);

		float Zenv = d_Zenv[idx];  // m

		float prevTenv = d_prevTenv[idx];  // K
		ASSERT(prevTenv > 100.);

		float prevPenv = d_prevPenv[idx];  // hPa
		ASSERT(prevPenv < 1200.);

		float prevZenv = d_prevZenv[idx];  // m

		float Tparcel = d_Tparcel[idx];  // K
		ASSERT(Tparcel > 100. || IsMissing(Tparcel));

		float prevTparcel = d_prevTparcel[idx];  // K
		ASSERT(prevTparcel > 100. || IsMissing(prevTparcel));

		const float LFCP = d_LFCP[idx];  // hPa
		const float LFCT = d_LFCT[idx];  // K

		if (IsMissing(Penv) || IsMissing(Tenv) || IsMissing(Zenv) || IsMissing(prevZenv) || IsMissing(Tparcel) ||
		    Penv > LFCP)
		{
			// Missing data or current grid point is below LFC
			return;
		}

		ASSERT(LFCP < 1200.);
		ASSERT(LFCT > 100.);

		if (IsMissing(prevTparcel) && !IsMissing(Tparcel))
		{
			// When rising above LFC, get accurate value of Tenv at that level so that even small amounts of CAPE
			// (and EL!) values can be determined.

			prevTenv = interpolation::Linear<float>(LFCP, prevPenv, Penv, prevTenv, Tenv);
			prevZenv = interpolation::Linear<float>(LFCP, prevPenv, Penv, prevZenv, Zenv);
			prevPenv = LFCP;     // LFC pressure
			prevTparcel = LFCT;  // LFC temperature

			// If LFC was found close to lower hybrid level, the linear interpolation and moist lift will result
			// to same values. In this case CAPE integration fails as there is no area formed between environment
			// and parcel temperature. The result for this is that LFC is found but EL is not found. To prevent
			// this, warm the parcel value just slightly so that a miniscule CAPE area is formed and EL is found.

			if (fabs(prevTparcel - prevTenv) < 0.0001)
			{
				prevTparcel += 0.0001;
			}
		}

		if (d_curLevel < d_breakLevel && (Tenv - Tparcel) > 25.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}
		else
		{
			if (prevZenv < 3000.)
			{
				float C = CAPE::CalcCAPE3km(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

				d_CAPE3km[idx] += C;

				ASSERT(d_CAPE3km[idx] >= 0);
			}

			float C = CAPE::CalcCAPE1040(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

			d_CAPE1040[idx] += C;

			ASSERT(d_CAPE1040[idx] >= 0);

			float CAPE, ELT, ELP, ELZ;
			CAPE::CalcCAPE(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv, CAPE, ELT, ELP, ELZ);

			d_CAPE[idx] += CAPE;

			ASSERT(CAPE >= 0.);

			if (IsValid(ELT))
			{
				if (IsMissing(d_ELT[idx]))
				{
					d_ELT[idx] = ELT;
				}

				d_ELP[idx] = fmaxf(ELP, d_ELP[idx]);
				d_ELZ[idx] = fminf(ELZ, d_ELZ[idx]);

				d_LastELT[idx] = ELT;
				d_LastELP[idx] = ELP;
				d_LastELZ[idx] = ELZ;
			}
		}
	}
}

__global__ void CINKernel(const float* __restrict__ d_Tenv, const float* __restrict__ d_prevTenv,
                          const float* __restrict__ d_Penv, const float* __restrict__ d_prevPenv,
                          const float* __restrict__ d_Zenv, const float* __restrict__ d_prevZenv,
                          const float* __restrict__ d_Tparcel, const float* __restrict__ d_prevTparcel,
                          const float* __restrict__ d_PLCL, const float* __restrict__ d_PLFC,
                          const float* __restrict__ d_Psource, float* __restrict__ d_cinh,
                          unsigned char* __restrict__ d_found, bool useVirtualTemperature, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N && d_found[idx] == 0)
	{
		float Tenv = d_Tenv[idx];  // K
		ASSERT(Tenv >= 150.);

		const float prevTenv = d_prevTenv[idx];

		float Penv = d_Penv[idx];  // hPa
		ASSERT(Penv < 1200. || IsMissing(Penv));

		const float prevPenv = d_prevPenv[idx];

		float Tparcel = d_Tparcel[idx];  // K
		ASSERT(Tparcel >= 150. || IsMissing(Tparcel));

		const float prevTparcel = d_prevTparcel[idx];

		const float PLFC = d_PLFC[idx];  // hPa
		ASSERT(PLFC < 1200. || IsMissing(PLFC));

		const float PLCL = d_PLCL[idx];  // hPa
		ASSERT(PLCL < 1200. || IsMissing(PLCL));

		float Zenv = d_Zenv[idx];          // m
		float prevZenv = d_prevZenv[idx];  // m

		// Make sure we have passed the starting level
		if (Penv <= d_Psource[idx])
		{
			if (Penv <= PLFC)
			{
				// reached max height
				d_found[idx] = 1;

				// Integrate the final piece from previous level to LFC level

				if (IsMissing(prevTparcel) || IsMissing(prevPenv) || IsMissing(prevTenv))
				{
					Tparcel = MissingFloat();  // unable to proceed with CIN integration
				}
				else
				{
					// First get LFC height in meters
					Zenv = interpolation::Linear<float>(PLFC, prevPenv, Penv, prevZenv, Zenv);

					// LFC environment temperature value
					Tenv = interpolation::Linear<float>(PLFC, prevPenv, Penv, prevTenv, Tenv);

					// LFC T parcel value
					Tparcel = interpolation::Linear<float>(PLFC, prevPenv, Penv, prevTparcel, Tparcel);

					Penv = PLFC;

					if (Zenv < prevZenv)
					{
						prevZenv = Zenv;
					}
				}
			}

			if (Penv < PLCL && useVirtualTemperature)
			{
				// Above LCL, switch to virtual temperature

				Tparcel = metutil::VirtualTemperature_<float>(Tparcel, Penv * 100);
				Tenv = metutil::VirtualTemperature_<float>(Tenv, Penv * 100);
			}

			if (!IsMissing(Tparcel))
			{
				d_cinh[idx] += CAPE::CalcCIN(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);
				ASSERT(d_cinh[idx] <= 0);
			}

			ASSERT(d_cinh[idx] <= 0.f);
		}
	}
}

__global__ void LFCKernel(const float* __restrict__ d_T, const float* __restrict__ d_P,
                          const float* __restrict__ d_prevT, const float* __restrict__ d_prevP,
                          float* __restrict__ d_Tparcel, const float* __restrict__ d_prevTparcel,
                          const float* __restrict__ d_LCLT, const float* __restrict__ d_LCLP,
                          float* __restrict__ d_LFCT, float* __restrict__ d_LFCP, float* __restrict__ d_LastLFCT,
                          float* __restrict__ d_LastLFCP, unsigned char* __restrict__ d_found, int d_curLevel,
                          int d_breakLevel, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N && d_found[idx] == 0)
	{
		float Tparcel = d_Tparcel[idx];
		float prevTparcel = d_prevTparcel[idx];
		float Tenv = d_T[idx];

		ASSERT(Tenv < 350.);
		ASSERT(Tenv > 100.);

		float prevTenv = d_prevT[idx];
		ASSERT(prevTenv < 350.);
		ASSERT(prevTenv > 100.);

		float Penv = d_P[idx];
		float prevPenv = d_prevP[idx];

		ASSERT(Penv > 50.);
		ASSERT(Penv < 1200.);
		float LCLP = d_LCLP[idx];
		ASSERT(prevPenv > 50.);
		ASSERT(prevPenv < 1200.);

		if ((d_curLevel < d_breakLevel && (Tenv - Tparcel) > 30.) || (IsValid(d_LFCT[idx]) && Penv < 650.))
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}

		const float diff = Tparcel - Tenv;
		const float prevdiff = prevTparcel - prevTenv;
		const bool isFirstLFC = (diff >= 0 || fabs(diff) < 1e-4) && IsMissing(prevdiff) && IsMissing(d_LFCT[idx]);
		const bool isLastLFC = (diff >= 0 || fabs(diff) < 1e-4) && prevdiff < 0;

		if (d_found[idx] == 0 && Penv < LCLP && (isFirstLFC || isLastLFC))
		{
			if (IsMissing(prevTparcel))
			{
				prevTparcel = d_LCLT[idx];  // previous is LCL
				ASSERT(!IsMissing(d_LCLT[idx]));
			}

			float& Tresult = (IsMissing(d_LFCT[idx])) ? d_LFCT[idx] : d_LastLFCT[idx];
			float& Presult = (IsMissing(d_LFCP[idx])) ? d_LFCP[idx] : d_LastLFCP[idx];

			if (diff < 0.01)
			{
				Tresult = Tparcel;
				Presult = Penv;
			}
			else if (prevTparcel - prevTenv >= 0)
			{
				Tresult = prevTparcel;
				Presult = prevPenv;
			}
			else
			{
				auto intersection = CAPE::GetPointOfIntersection(point(Tenv, Penv), point(prevTenv, prevPenv),
				                                                 point(Tparcel, Penv), point(prevTparcel, prevPenv));

				Tresult = intersection.X();
				Presult = intersection.Y();

				if (Presult > prevPenv)
				{
					// Do not allow LFC to be below previous level; if intersection fails to put it in the correct
					// "bin" (between previous and current pressure), use the only information that certain:
					// the crossing has happened at least at current pressure
					Tresult = Tparcel;
					Presult = Penv;
				}
				else if (IsMissing(Tresult))
				{
					// Intersection not found, use exact level value
					Tresult = Tparcel;
					Presult = Penv;
				}
			}

			ASSERT(Tresult > 100);
			ASSERT(Tresult < 350);
		}
	}
}

__global__ void ThetaEKernel(float* __restrict__ d_T, const float* __restrict__ d_RH, float* __restrict__ d_P,
                             const float* __restrict__ d_prevT, const float* __restrict__ d_prevRH,
                             const float* __restrict__ d_prevP, float* __restrict__ d_ThetaE, float* __restrict__ d_TD,
                             unsigned char* __restrict__ d_found, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		float ThetaE = MissingFloat(), TD = MissingFloat();

		if (d_found[idx] == 0)
		{
			float& T = d_T[idx];
			float& P = d_P[idx];
			float RH = d_RH[idx];

			if (P < mucape_search_limit)
			{
				T = interpolation::Linear<float>(mucape_search_limit, P, d_prevP[idx], T, d_prevT[idx]);
				RH = interpolation::Linear<float>(mucape_search_limit, P, d_prevP[idx], RH, d_prevRH[idx]);

				d_found[idx] = 1;  // Make sure this is the last time we access this grid point
				P = mucape_search_limit;
			}

			TD = metutil::DewPointFromRH_<float>(T, RH);
			ThetaE = metutil::smarttool::ThetaE_<float>(T, RH, P * 100);
		}

		d_ThetaE[idx] = ThetaE;
		d_TD[idx] = TD;
	}
}

__global__ void MixingRatioKernel(const __half* __restrict__ d_T, const float* __restrict__ d_Pstart,
                                  const float* __restrict__ d_Pstop, const __half* __restrict__ d_RH,
                                  const float* __restrict__ d_Tfirst, const float* __restrict__ d_RHfirst,
                                  float* __restrict__ d_Tpot, float* __restrict__ d_MR, size_t k, size_t n, size_t N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T);
	ASSERT(d_RH);
	ASSERT(d_Pstart);

	if (idx < N)
	{
		const float T = d_Tfirst[idx] - __half2float(d_T[idx * n + k]);
		float P = d_Pstart[idx] - 1.f * k;

		if (P < d_Pstop[idx])
		{
			P = himan::MissingFloat();
		}

		const float RH = d_RHfirst[idx] - __half2float(d_RH[idx * n + k]);

		ASSERT((T > 150 && T < 350) || IsMissing(T));
		ASSERT((P > 100 && P < 1500) || IsMissing(P));
		ASSERT((RH >= 0 && RH < 102) || IsMissing(RH));

		d_Tpot[idx] = metutil::Theta_<float>(T, 100 * P);
		d_MR[idx] = metutil::smarttool::MixingRatio_<float>(T, RH, 100 * P);
	}
}

__global__ void MixingRatioFinalizeKernel(float* __restrict__ d_T, float* __restrict__ d_TD,
                                          const float* __restrict__ d_P, const float* __restrict__ d_Tpot,
                                          const float* __restrict__ d_MR, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const float P = d_P[idx];
		const float MR = d_MR[idx];
		const float Tpot = d_Tpot[idx];

		ASSERT((P > 100 && P < 1500) || IsMissing(P));

		float T = Tpot * pow((P / 1000.), 0.2854);
		const float Es = metutil::Es_<float>(T);  // Saturated water vapor pressure
		const float E = metutil::E_<float>(MR, 100 * P);
		const float RH = fminf(102., E / Es * 100);

		d_TD[idx] = metutil::DewPointFromRH_<float>(T, RH);
		d_T[idx] = T;

		if (isnan(T) || isnan(d_TD[idx]))
		{
			// half is mangling the missing value-nan
			d_T[idx] = himan::MissingFloat();
			d_TD[idx] = himan::MissingFloat();
		}
	}
}

__global__ void Max1D(const float* __restrict__ d_v, unsigned char* __restrict__ d_maxima, unsigned char mask_len,
                      unsigned char K, size_t N)
{
	ASSERT(mask_len % 2 == 1);

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const unsigned char half = mask_len / 2;

		// data layout is changed here wrt to the source

		// old layout:
		// |x(0)y(0)z(0)..n(0)|x(1)y(1)z(1)..n(1)|..|x(N)y(N)z(N)..n(N)|

		// new layout:
		// |x(0)x(1)x(2)..x(K)|y(1)y(2)y(2)..y(K)|..|n(1)n(1)n(3)..n(K)|

		// beginning

		for (unsigned char i = 0; i < half; i++)
		{
			float maxv = d_v[idx];
			unsigned char maxl = 0;  // first guess

			for (unsigned char j = 1; j <= half + i; j++)
			{
				if (d_v[idx + j * N] > maxv)
				{
					maxv = d_v[idx + j * N];
					maxl = j;
				}
			}
			d_maxima[i + idx * K] = maxl;
		}

		// center

		for (unsigned char i = half; i < K - half; i++)
		{
			float maxv = d_v[idx + (i - half) * N];
			unsigned char maxl = i - half;

			for (unsigned char j = i - half + 1; j <= i + half; j++)
			{
				if (d_v[idx + j * N] > maxv)
				{
					maxv = d_v[idx + j * N];
					maxl = j;
				}
			}
			d_maxima[i + idx * K] = maxl;
		}

		// end

		for (unsigned char i = K - half; i < K; i++)
		{
			float maxv = d_v[idx + (i - half) * N];
			unsigned char maxl = i - half;

			for (unsigned char j = i - half + 1; j < K; j++)
			{
				if (d_v[idx + j * N] > maxv)
				{
					maxv = d_v[idx + j * N];
					maxl = j;
				}
			}
			d_maxima[i + idx * K] = maxl;
		}
	}
}

__global__ void MaximaLocation(const float* __restrict__ d_v, const unsigned char* __restrict__ d_maxima,
                               unsigned char* __restrict__ d_idx, size_t K, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const int maxMax = K / 4;

		int maximaN = 0;

		for (int i = 0; i < K && maximaN < (maxMax - 1); i++)
		{
			const float v = d_v[idx + i * N];  // ThetaE value at this point in the profile

			if (i == d_maxima[i + idx * K])
			{
				if (i > 0 && v == d_v[idx + (i - 1) * N])
				{
					// Duplicate maximas (two consecutive vertical levels
					// have the same thetae value and are both maximas).
					// Disregard this higher one.
				}
				else
				{
					d_idx[maximaN + 1 + idx * maxMax] = i;
					maximaN++;
				}
			}
		}

		d_idx[idx * maxMax] = maximaN;

		// bubble sort: highest value theta e should be first

		bool passed;

		do
		{
			passed = true;

			for (int i = 2; i < maximaN + 1; i++)
			{
				unsigned char& previ = d_idx[i - 1 + idx * maxMax];
				unsigned char& curi = d_idx[i + idx * maxMax];
				float prev = d_v[previ * N + idx];
				float cur = d_v[curi * N + idx];

				if (prev < cur)
				{
					unsigned char tmpi = curi;
					curi = previ;
					previ = tmpi;

					passed = false;
				}
			}
		} while (!passed);
	}
}

__global__ void MeanKernel(const float* __restrict__ d_Tpot, const float* __restrict__ d_MR,
                           const float* __restrict__ d_prevTpot, const float* __restrict__ d_prevMR,
                           float* __restrict__ d_meanTpot, float* __restrict__ d_meanMR, float* __restrict__ d_range,
                           size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		if (IsValid(d_Tpot[idx]))
		{
			// trapezoidal integration
			d_range[idx] += 1;
			d_meanTpot[idx] += (d_prevTpot[idx] + d_Tpot[idx]) * 0.5;
			d_meanMR[idx] += (d_prevMR[idx] + d_MR[idx]) * 0.5;
		}
	}
}

__global__ void MeanFinalizeKernel(float* __restrict__ d_meanTpot, float* __restrict__ d_meanMR,
                                   const float* __restrict__ d_range, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		d_meanTpot[idx] = d_meanTpot[idx] / d_range[idx];
		d_meanMR[idx] = d_meanMR[idx] / d_range[idx];
	}
}

__global__ void CopyProfileValuesKernel(const float* __restrict__ d_first, __half* __restrict__ d_profile,
                                        const float* __restrict__ d_arr, int i, size_t n, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		d_profile[idx * n + i] = __float2half(d_first[idx] - d_arr[idx]);
	}
}

__global__ void SampleKernel(const __half* __restrict__ d_x, const __half* __restrict__ d_y,
                             const float* __restrict__ d_yfirst, const float* __restrict__ d_x0,
                             const float* __restrict__ d_x1, __half* __restrict__ d_sampled, float* d_first,
                             size_t levelCount, size_t sampleCount, size_t N)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		// because lowest profile level is not equal to starting level,
		// we might have the case where starting level is outside the
		// profile
		int h = 0;

		for (int i = 0; i < sampleCount; i++)
		{
			float sample = d_x0[idx] - static_cast<float>(i);

			if (sample < d_x1[idx])
			{
				sample = himan::MissingFloat();
			}

			d_sampled[idx * sampleCount + i] = __float2half(himan::MissingFloat());

			for (int j = 1; j < levelCount; j++)
			{
				const int ii = idx * levelCount + j - 1;
				const float x1 = d_x0[idx] - __half2float(d_x[ii]);
				const float y1 = d_yfirst[idx] - __half2float(d_y[ii]);
				const float x2 = d_x0[idx] - __half2float(d_x[ii + 1]);
				const float y2 = d_yfirst[idx] - __half2float(d_y[ii + 1]);

				if (x1 >= sample && x2 <= sample)
				{
					const float val = interpolation::Linear<float>(sample, x1, x2, y1, y2);

					if (h == 0)
					{
						d_first[idx] = val;
					}

					d_sampled[idx * sampleCount + h] = __float2half(d_first[idx] - val);
					h++;
					break;
				}
			}
		}
	}
}

cape_multi_source cape_cuda::GetNHighestThetaEValuesGPU(const std::shared_ptr<const plugin_configuration>& conf,
                                                        std::shared_ptr<info<float>> myTargetInfo, int n)
{
	himan::level curLevel = itsBottomLevel;

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	// We need to get the number of layers so we can preallocate
	// a suitable sized array.

	const auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), mucape_search_limit, conf->TargetGeomName());
	const auto levelSpan = curLevel.Value() - stopLevel.second.Value();

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_T = 0;
	float* d_TD = 0;
	float* d_P = 0;
	float* d_ThetaE = 0;
	float* d_RH = 0;
	float* d_prevT = 0;
	float* d_prevP = 0;
	float* d_prevRH = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((float**)&d_T, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_TD, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_P, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ThetaE, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((float**)&d_RH, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevRH, sizeof(float) * N));

	InitializeArray<float>(d_T, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_TD, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_P, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevRH, himan::MissingFloat(), N, stream);

	InitializeArray<unsigned char>(d_found, 0, N, stream);

	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	// profiles are create as flattened vectors
	// in order the insertion to be as fast as possible, the layout is such:
	// |x(0)y(0)z(0)..n(0)|x(1)y(1)z(1)..n(1)|..|x(N)y(N)z(N)..n(N)|

	std::vector<float> ThetaEProfile(levelSpan * N), TProfile(levelSpan * N), PProfile(levelSpan * N),
	    TDProfile(levelSpan * N);

	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(ThetaEProfile.data()), sizeof(float) * levelSpan * N, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(TProfile.data()), sizeof(float) * N * levelSpan, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(TDProfile.data()), sizeof(float) * N, 0));
	CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(PProfile.data()), sizeof(float) * N, 0));

	size_t K = 0;  // this will hold the number of levels read (should match what we calculated previously)
	while (true)
	{
		auto TInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		auto RHInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, RHParam, myTargetInfo->ForecastType());
		auto PInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());

		if (!TInfo || !RHInfo || !PInfo)
		{
			CUDA_CHECK(hipHostUnregister(ThetaEProfile.data()));
			CUDA_CHECK(hipHostUnregister(TProfile.data()));
			CUDA_CHECK(hipHostUnregister(TDProfile.data()));
			CUDA_CHECK(hipHostUnregister(PProfile.data()));

			CUDA_CHECK(hipFree(d_T));
			CUDA_CHECK(hipFree(d_P));
			CUDA_CHECK(hipFree(d_RH));
			CUDA_CHECK(hipFree(d_prevT));
			CUDA_CHECK(hipFree(d_prevP));
			CUDA_CHECK(hipFree(d_prevRH));
			CUDA_CHECK(hipFree(d_ThetaE));
			CUDA_CHECK(hipFree(d_TD));
			CUDA_CHECK(hipFree(d_found));

			return cape_multi_source();
		}

		hc::PrepareInfo(TInfo, d_T, stream, conf->UseCacheForReads());
		hc::PrepareInfo(PInfo, d_P, stream, conf->UseCacheForReads());
		hc::PrepareInfo(RHInfo, d_RH, stream, conf->UseCacheForReads());

		ThetaEKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_RH, d_P, d_prevT, d_prevRH, d_prevP, d_ThetaE, d_TD,
		                                                 d_found, N);

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);

		CUDA_CHECK(hipMemcpyAsync(&ThetaEProfile[K * N], d_ThetaE, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&TProfile[K * N], d_T, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&TDProfile[K * N], d_TD, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&PProfile[K * N], d_P, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));

		curLevel.Value(curLevel.Value() - 1);
		K++;

		if (foundCount == N || levelSpan == K)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevT, d_T, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevP, d_P, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevRH, d_RH, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
	}

	CUDA_CHECK(hipHostUnregister(ThetaEProfile.data()));
	CUDA_CHECK(hipHostUnregister(TProfile.data()));
	CUDA_CHECK(hipHostUnregister(TDProfile.data()));
	CUDA_CHECK(hipHostUnregister(PProfile.data()));

	CUDA_CHECK(hipFree(d_T));
	CUDA_CHECK(hipFree(d_P));
	CUDA_CHECK(hipFree(d_RH));
	CUDA_CHECK(hipFree(d_prevT));
	CUDA_CHECK(hipFree(d_prevP));
	CUDA_CHECK(hipFree(d_prevRH));
	CUDA_CHECK(hipFree(d_ThetaE));
	CUDA_CHECK(hipFree(d_TD));
	CUDA_CHECK(hipFree(d_found));

	// Check comments from cape.cpp
	vec2d Tret(n), TDret(n), Pret(n);

	for (size_t j = 0; j < static_cast<size_t>(n); j++)
	{
		Tret[j].resize(N, MissingFloat());
		TDret[j].resize(N, MissingFloat());
		Pret[j].resize(N, MissingFloat());
	}

	float* d_v = 0;
	unsigned char* d_maxima = 0;
	unsigned char* d_idxs = 0;

	CUDA_CHECK(hipMalloc((float**)&d_v, sizeof(float) * N * K));  // Actual ThetaE values
	CUDA_CHECK(hipMalloc((unsigned char**)&d_maxima,
	                      sizeof(unsigned char) * N * K));  // Local maxima locations in the profile

	CUDA_CHECK(hipMemcpyAsync(d_v, ThetaEProfile.data(), sizeof(float) * N * K, hipMemcpyHostToDevice, stream));

	Max1D<<<gridSize, blockSize, 0, stream>>>(d_v, d_maxima, 5, K, N);

	// maximum number of maximas we expect to find in the profile
	const size_t maxMax = K / 4;

	CUDA_CHECK(hipMalloc((unsigned char**)&d_idxs, sizeof(unsigned char) * N * maxMax));

	MaximaLocation<<<gridSize, blockSize, 0, stream>>>(d_v, d_maxima, d_idxs, K, N);

	std::vector<unsigned char> idxs(N * maxMax);

	CUDA_CHECK(hipMemcpyAsync(&idxs[0], d_idxs, sizeof(unsigned char) * N * maxMax, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_v));
	CUDA_CHECK(hipFree(d_maxima));
	CUDA_CHECK(hipFree(d_idxs));

	for (size_t i = 0; i < N; i++)
	{
		const size_t s = i * maxMax;     // start index of this grid point
		const size_t maximaN = idxs[s];  // number of maximas found (at most maxMax)

		ASSERT(maximaN > 0);
		ASSERT(maximaN <= maxMax);

		// Remove maximas that are too high in the atmosphere
		size_t newMaximaN = 0;
		size_t offset = 0;
		for (size_t j = 0; j < maximaN; j++)
		{
			const size_t sidx = 1 + s + j;            // index in the array where maxima index is found
			const unsigned char maxidx = idxs[sidx];  // index in the vertical profile where the maxima was found

			if (PProfile[maxidx * N + i] < mucape_maxima_search_limit)
			{
				offset++;
				continue;
			}
			newMaximaN++;
			idxs[sidx - offset] = maxidx;
		}

		for (size_t j = 0; j < min(static_cast<size_t>(n), newMaximaN); j++)
		{
			const size_t sidx = 1 + s + j;  // index in the array where maxima index is found
			const short maxidx =
			    static_cast<short>(idxs[sidx]);  // index in the vertical profile where the maxima was found

			ASSERT(static_cast<unsigned>(maxidx) <= K);

			// Copy values from max theta e levels for further processing

			Tret[j][i] = TProfile[maxidx * N + i];
			TDret[j][i] = TDProfile[maxidx * N + i];
			Pret[j][i] = PProfile[maxidx * N + i];

			ASSERT(IsValid(Tret[j][i]));
			ASSERT(IsValid(TDret[j][i]));
			ASSERT(IsValid(Pret[j][i]));
		}
	}

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_tuple(Tret, TDret, Pret);
}

void GetSampledSourceDataGPU(std::shared_ptr<const himan::plugin_configuration> conf,
                             std::shared_ptr<himan::info<float>> myTargetInfo, const float* d_P500m,
                             const float* d_Psurface, const float* d_Tsurface, const float* d_RHsurface,
                             __half* d_temperatureSample, __half* d_humiditySample, float* __restrict__ d_Tfirst,
                             float* __restrict__ d_RHfirst, const himan::level& startLevel,
                             const himan::level& stopLevel, unsigned int sampleCount, hipStream_t& stream)
{
	using namespace himan;
	const size_t N = myTargetInfo->SizeLocations();
	level curLevel = startLevel;

	const int levelCount = 1 + static_cast<int>(curLevel.Value() - stopLevel.Value());
	__half* d_pressureProfile = 0;
	__half* d_temperatureProfile = 0;
	__half* d_humidityProfile = 0;
	float* d_arr = 0;
	CUDA_CHECK(hipMalloc((__half**)&d_temperatureProfile, N * levelCount * sizeof(__half)));
	CUDA_CHECK(hipMalloc((__half**)&d_pressureProfile, N * levelCount * sizeof(__half)));
	CUDA_CHECK(hipMalloc((__half**)&d_humidityProfile, N * levelCount * sizeof(__half)));
	CUDA_CHECK(hipMalloc((float**)&d_arr, N * sizeof(float)));

	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	unsigned k = 0;

	while (curLevel.Value() >= stopLevel.Value())
	{
		auto TInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		hc::PrepareInfo(TInfo, d_arr, stream, conf->UseCacheForReads());
		CopyProfileValuesKernel<<<gridSize, blockSize, 0, stream>>>(d_Tsurface, d_temperatureProfile, d_arr, k,
		                                                            levelCount, N);

		auto PInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());
		hc::PrepareInfo(PInfo, d_arr, stream, conf->UseCacheForReads());
		CopyProfileValuesKernel<<<gridSize, blockSize, 0, stream>>>(d_Psurface, d_pressureProfile, d_arr, k, levelCount,
		                                                            N);

		auto RHInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, RHParam, myTargetInfo->ForecastType());
		hc::PrepareInfo(RHInfo, d_arr, stream, conf->UseCacheForReads());
		CopyProfileValuesKernel<<<gridSize, blockSize, 0, stream>>>(d_RHsurface, d_humidityProfile, d_arr, k,
		                                                            levelCount, N);

		k++;
		curLevel.Value(curLevel.Value() - 1);
	}

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_arr));

	SampleKernel<<<gridSize, blockSize, 0, stream>>>(d_pressureProfile, d_temperatureProfile, d_Tsurface, d_Psurface,
	                                                 d_P500m, d_temperatureSample, d_Tfirst, levelCount, sampleCount,
	                                                 N);
	SampleKernel<<<gridSize, blockSize, 0, stream>>>(d_pressureProfile, d_humidityProfile, d_RHsurface, d_Psurface,
	                                                 d_P500m, d_humiditySample, d_RHfirst, levelCount, sampleCount, N);

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_pressureProfile));
	CUDA_CHECK(hipFree(d_temperatureProfile));
	CUDA_CHECK(hipFree(d_humidityProfile));
}

cape_source cape_cuda::Get500mMixingRatioValuesGPU(std::shared_ptr<const plugin_configuration>& conf,
                                                   std::shared_ptr<info<float>> myTargetInfo)
{
	myTargetInfo->FirstValidGrid();
	const size_t N = myTargetInfo->Data().Size();

	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	level curLevel = itsBottomLevel;

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());

	auto PInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType(), false);
	auto TInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType(), false);
	auto RHInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, RHParam, myTargetInfo->ForecastType(), false);

	if (!PInfo || PInfo->Data().MissingCount() == PInfo->SizeLocations() || !TInfo || !RHInfo)
	{
		return std::make_tuple(std::vector<float>(), std::vector<float>(), std::vector<float>());
	}

	auto PSurface = VEC(PInfo);
	auto TSurface = VEC(TInfo);
	auto RHSurface = VEC(RHInfo);

	auto P500m = h->VerticalValue<double>(PParam, 500.);
	auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), 500., conf->TargetGeomName());

	auto P500mf = util::Convert<double, float>(P500m);

	float* d_Psurface = 0;
	float* d_P500m = 0;
	float* d_Tsurface = 0;
	float* d_RHsurface = 0;
	float* d_Tfirst = 0;
	float* d_RHfirst = 0;

	CUDA_CHECK(hipMalloc((float**)&d_Psurface, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_P500m, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tsurface, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_RHsurface, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tfirst, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_RHfirst, N * sizeof(float)));

	CUDA_CHECK(hipMemcpyAsync(d_Psurface, PSurface.data(), N * sizeof(float), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_P500m, P500mf.data(), N * sizeof(float), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Tsurface, TSurface.data(), N * sizeof(float), hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_RHsurface, RHSurface.data(), N * sizeof(float), hipMemcpyHostToDevice, stream));

	// maximum number of samples is much more than number of read levels
	// for every 1hPa step we need one sample
	unsigned int sampleCount = 0;
	for (size_t i = 0; i < N; i++)
	{
		sampleCount = max(sampleCount, static_cast<unsigned int>(ceil(PSurface[i] - P500mf[i])));
	}

	// Use half-precision data type to store the samples and the vertical profile from which the samples
	// are created (that's in function GetSampledSourceDataGPU()).
	//
	// 16-bit floating precision data type allows only 1024 distinct values between 2e-14 ... 2e15. That
	// is too little to provide reasonable loss of accuracy for us. Therefore in the half-profiles and samples
	// we store the offset of the sampled variable relative to some base value. This enables us to store smaller
	// values (in absolute magnitude) and therefore allow more precision.
	//
	// The original value can be restored using the base value (32-bit) and offset (16-bit), ie. each sampled
	// value is a direct derivative from the base. Another alternative would have been to store each sample as and
	// offset of the previous value: this would have given even smaller values (=more precision), but code-wise
	// calculating the original value would be more complicated as when recovering value for sample order number
	// x we'd need to access all values between base and x.

	__half* d_temperatureSample = 0;
	__half* d_humiditySample = 0;

	CUDA_CHECK(hipMalloc((__half**)&d_humiditySample, N * sampleCount * sizeof(__half)));
	CUDA_CHECK(hipMalloc((__half**)&d_temperatureSample, N * sampleCount * sizeof(__half)));

	GetSampledSourceDataGPU(conf, myTargetInfo, d_P500m, d_Psurface, d_Tsurface, d_RHsurface, d_temperatureSample,
	                        d_humiditySample, d_Tfirst, d_RHfirst, itsBottomLevel, stopLevel.second, sampleCount,
	                        stream);

	float* d_Tpot = 0;
	float* d_MR = 0;
	float* d_prevTpot = 0;
	float* d_prevMR = 0;
	float* d_meanTpot = 0;
	float* d_meanMR = 0;
	float* d_range = 0;

	CUDA_CHECK(hipMalloc((float**)&d_Tpot, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_MR, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevTpot, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevMR, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_meanTpot, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_meanMR, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_range, N * sizeof(float)));

	InitializeArray<float>(d_meanTpot, 0.f, N, stream);
	InitializeArray<float>(d_meanMR, 0.f, N, stream);
	InitializeArray<float>(d_range, 0.f, N, stream);

	for (unsigned int k = 0; k < sampleCount; k++)
	{
		MixingRatioKernel<<<gridSize, blockSize, 0, stream>>>(d_temperatureSample, d_Psurface, d_P500m,
		                                                      d_humiditySample, d_Tfirst, d_RHfirst, d_Tpot, d_MR, k,
		                                                      sampleCount, N);
		if (k > 0)
		{
			MeanKernel<<<gridSize, blockSize, 0, stream>>>(d_Tpot, d_MR, d_prevTpot, d_prevMR, d_meanTpot, d_meanMR,
			                                               d_range, N);
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevTpot, d_Tpot, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevMR, d_MR, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
	}

	MeanFinalizeKernel<<<gridSize, blockSize, 0, stream>>>(d_meanTpot, d_meanMR, d_range, N);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_temperatureSample));
	CUDA_CHECK(hipFree(d_humiditySample));
	CUDA_CHECK(hipFree(d_Tpot));
	CUDA_CHECK(hipFree(d_MR));
	CUDA_CHECK(hipFree(d_prevTpot));
	CUDA_CHECK(hipFree(d_prevMR));
	CUDA_CHECK(hipFree(d_range));
	CUDA_CHECK(hipFree(d_P500m));
	CUDA_CHECK(hipFree(d_Tsurface));
	CUDA_CHECK(hipFree(d_RHsurface));
	CUDA_CHECK(hipFree(d_Tfirst));
	CUDA_CHECK(hipFree(d_RHfirst));

	float* d_T = 0;
	float* d_TD = 0;

	CUDA_CHECK(hipMalloc((float**)&d_TD, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_T, N * sizeof(float)));

	std::vector<float> TD(N);
	std::vector<float> T(N);

	MixingRatioFinalizeKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_TD, d_Psurface, d_meanTpot, d_meanMR, N);

	CUDA_CHECK(hipMemcpyAsync(T.data(), d_T, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(TD.data(), d_TD, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_meanTpot));
	CUDA_CHECK(hipFree(d_meanMR));
	CUDA_CHECK(hipFree(d_T));
	CUDA_CHECK(hipFree(d_TD));
	CUDA_CHECK(hipFree(d_Psurface));

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_tuple(T, TD, PSurface);
}

std::vector<std::pair<std::vector<float>, std::vector<float>>> cape_cuda::GetLFCGPU(
    const std::shared_ptr<const plugin_configuration>& conf, std::shared_ptr<info<float>> myTargetInfo,
    std::vector<float>& T, std::vector<float>& P, std::vector<float>& TenvLCL)
{
	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_LCLP = 0;
	float* d_LCLT = 0;
	float* d_LFCT = 0;
	float* d_LFCP = 0;
	float* d_LastLFCT = 0;
	float* d_LastLFCP = 0;
	float* d_Tparcel = 0;
	float* d_prevTparcel = 0;
	float* d_Tenv = 0;
	float* d_Penv = 0;
	float* d_prevTenv = 0;
	float* d_prevPenv = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((float**)&d_LCLT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LCLP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LFCT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LFCP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastLFCT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastLFCP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Penv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevPenv, sizeof(float) * N));

	CUDA_CHECK(hipMemcpyAsync(d_LCLT, &T[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LCLP, &P[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));

	InitializeArray<float>(d_LFCT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LFCP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastLFCT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastLFCP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_prevTparcel, himan::MissingFloat(), N, stream);
	InitializeArray<unsigned char>(d_found, 0, N, stream);

	// For each grid point find the hybrid level that's below LCL and then pick the lowest level
	// among all grid points; most commonly it's the lowest hybrid level

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P), conf->TargetGeomName());

	level curLevel = levels.first;

	auto prevPenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());
	auto prevTenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());

	hc::PrepareInfo(prevTenvInfo, d_prevTenv, stream, conf->UseCacheForReads());
	hc::PrepareInfo(prevPenvInfo, d_prevPenv, stream, conf->UseCacheForReads());

	if (cape_cuda::itsUseVirtualTemperature)
	{
		VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_prevTenv, d_prevPenv, N);
	}

	curLevel.Value(curLevel.Value() - 1);

	std::vector<unsigned char> found(N, 0);
	std::vector<float> LFCT(N, himan::MissingFloat());
	std::vector<float> LFCP(N, himan::MissingFloat());
	std::vector<float> LastLFCT(N);
	std::vector<float> LastLFCP(N);

	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	for (size_t i = 0; i < N; i++)
	{
		if ((T[i] - TenvLCL[i]) > 0.0001)
		{
			LFCT[i] = T[i];
			LFCP[i] = P[i];
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCT, &LFCT[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCP, &LFCP[0], sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	auto hPa450 = h->LevelForHeight(myTargetInfo->Producer(), 450., conf->TargetGeomName());
	auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), 250., conf->TargetGeomName());

	while (curLevel.Value() > stopLevel.first.Value())
	{
		// Get environment temperature and pressure values for this level
		auto TenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		auto PenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());

		hc::PrepareInfo(PenvInfo, d_Penv, stream, conf->UseCacheForReads());
		hc::PrepareInfo(TenvInfo, d_Tenv, stream, conf->UseCacheForReads());

		if (cape_cuda::itsUseVirtualTemperature)
		{
			VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, N);
		}

		// Lift the particle from previous level to this level. In the first revolution
		// of this loop the starting level is LCL. If target level level is below current level
		// (ie. we would be lowering the particle) missing value is returned.

		MoistLiftKernel<<<gridSize, blockSize, 0, stream>>>(d_LCLT, d_LCLP, d_Penv, d_Tparcel, N);

		LFCKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, d_prevTenv, d_prevPenv, d_Tparcel, d_prevTparcel,
		                                              d_LCLT, d_LCLP, d_LFCT, d_LFCP, d_LastLFCT, d_LastLFCP, d_found,
		                                              curLevel.Value(), hPa450.first.Value(), N);

		CUDA_CHECK(hipStreamSynchronize(stream));

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (N == foundCount)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTenv, d_Tenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevPenv, d_Penv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		curLevel.Value(curLevel.Value() - 1);
	}

	LastLFCCopyKernel<<<gridSize, blockSize, 0, stream>>>(d_LFCT, d_LFCP, d_LastLFCT, d_LastLFCP, N);

	CUDA_CHECK(hipMemcpyAsync(LFCT.data(), d_LFCT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(LFCP.data(), d_LFCP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(LastLFCT.data(), d_LastLFCT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(LastLFCP.data(), d_LastLFCP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipFree(d_LCLT));
	CUDA_CHECK(hipFree(d_LCLP));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_Penv));
	CUDA_CHECK(hipFree(d_Tenv));
	CUDA_CHECK(hipFree(d_prevPenv));
	CUDA_CHECK(hipFree(d_prevTenv));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_LastLFCT));
	CUDA_CHECK(hipFree(d_LastLFCP));

	CUDA_CHECK(hipStreamDestroy(stream));

	return {std::make_pair(LFCT, LFCP), std::make_pair(LastLFCT, LastLFCP)};
}

std::vector<float> cape_cuda::GetCINGPU(const std::shared_ptr<const plugin_configuration>& conf,
                                        std::shared_ptr<info<float>> myTargetInfo, const std::vector<float>& Tsource,
                                        const std::vector<float>& Psource, const std::vector<float>& PLCL,
                                        const std::vector<float>& PLFC, const std::vector<float>& ZLFC)
{
	const params PParams({param("PGR-PA"), param("P-PA")});

	forecast_time ftime = myTargetInfo->Time();
	forecast_type ftype = myTargetInfo->ForecastType();

	/*
	 * Modus operandi:
	 *
	 * 1. Integrate from ground to LCL dry adiabatically
	 *
	 * This can be done always since LCL is known at all grid points
	 * (that have source data values defined).
	 *
	 * 2. Integrate from LCL to LFC moist adiabatically
	 *
	 * Note! For some points integration will fail (no LFC found)
	 *
	 * We stop integrating at first time CAPE area is found!
	 */

	level curLevel = itsBottomLevel;

	auto prevZenvInfo = hc::Fetch<float>(conf, ftime, curLevel, ZParam, ftype);
	auto prevTenvInfo = hc::Fetch<float>(conf, ftime, curLevel, TParam, ftype);
	auto prevPenvInfo = hc::Fetch<float>(conf, ftime, curLevel, PParam, ftype);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_Psource = 0;
	float* d_Tparcel = 0;
	float* d_prevTparcel = 0;
	float* d_Tsource = 0;
	float* d_PLCL = 0;
	float* d_PLFC = 0;
	float* d_cinh = 0;
	float* d_prevZenv = 0;
	float* d_prevTenv = 0;
	float* d_prevPenv = 0;
	float* d_Zenv = 0;
	float* d_Tenv = 0;
	float* d_Penv = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((float**)&d_Psource, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tparcel, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevTparcel, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tsource, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_PLCL, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_PLFC, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_cinh, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevZenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevTenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_prevPenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Zenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Tenv, N * sizeof(float)));
	CUDA_CHECK(hipMalloc((float**)&d_Penv, N * sizeof(float)));

	CUDA_CHECK(hipMalloc((unsigned char**)&d_found, N * sizeof(unsigned char)));

	hc::PrepareInfo(prevZenvInfo, d_prevZenv, stream, conf->UseCacheForReads());
	hc::PrepareInfo(prevTenvInfo, d_prevTenv, stream, conf->UseCacheForReads());
	hc::PrepareInfo(prevPenvInfo, d_prevPenv, stream, conf->UseCacheForReads());

	InitializeArray<float>(d_cinh, 0., N, stream);
	InitializeArray<float>(d_Tparcel, himan::MissingFloat(), N, stream);

	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, Tsource.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Psource, Psource.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Tsource, d_prevTparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLCL, PLCL.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLFC, PLFC.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream));

	std::vector<unsigned char> found(N, 0);

	for (size_t i = 0; i < PLFC.size(); i++)
	{
		if (IsMissing(PLFC[i]))
		{
			found[i] = true;
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));

	curLevel.Value(curLevel.Value() - 1);

	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	auto hPa100 = h->LevelForHeight(myTargetInfo->Producer(), 100., conf->TargetGeomName());
	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	while (curLevel.Value() > hPa100.first.Value())
	{
		auto ZenvInfo = hc::Fetch<float>(conf, ftime, curLevel, ZParam, ftype);
		auto TenvInfo = hc::Fetch<float>(conf, ftime, curLevel, TParam, ftype);
		auto PenvInfo = hc::Fetch<float>(conf, ftime, curLevel, PParam, ftype);

		hc::PrepareInfo(ZenvInfo, d_Zenv, stream, conf->UseCacheForReads());
		hc::PrepareInfo(PenvInfo, d_Penv, stream, conf->UseCacheForReads());
		hc::PrepareInfo(TenvInfo, d_Tenv, stream, conf->UseCacheForReads());

		LiftLCLKernel<<<gridSize, blockSize, 0, stream>>>(d_Psource, d_Tsource, d_PLCL, d_Penv, d_Tparcel, N);

		CINKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_prevTenv, d_Penv, d_prevPenv, d_Zenv, d_prevZenv,
		                                              d_Tparcel, d_prevTparcel, d_PLCL, d_PLFC, d_Psource, d_cinh,
		                                              d_found, cape_cuda::itsUseVirtualTemperature, N);

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);
		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (N == foundCount)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevZenv, d_Zenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTenv, d_Tenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevPenv, d_Penv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		curLevel.Value(curLevel.Value() - 1);
	}

	std::vector<float> cinh(N, 0);

	CUDA_CHECK(hipMemcpyAsync(&cinh[0], d_cinh, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipFree(d_Psource));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_Tsource));
	CUDA_CHECK(hipFree(d_PLCL));
	CUDA_CHECK(hipFree(d_PLFC));
	CUDA_CHECK(hipFree(d_prevZenv));
	CUDA_CHECK(hipFree(d_prevPenv));
	CUDA_CHECK(hipFree(d_prevTenv));
	CUDA_CHECK(hipFree(d_Zenv));
	CUDA_CHECK(hipFree(d_Penv));
	CUDA_CHECK(hipFree(d_Tenv));
	CUDA_CHECK(hipFree(d_found));

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_cinh));

	CUDA_CHECK(hipStreamDestroy(stream));

	return cinh;
}

CAPEdata cape_cuda::GetCAPEGPU(const std::shared_ptr<const plugin_configuration>& conf,
                               std::shared_ptr<info<float>> myTargetInfo, const std::vector<float>& T,
                               const std::vector<float>& P)
{
	ASSERT(T.size() == P.size());

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	// Typically LFC value is not found to all grid points -- in many cases as few as
	// 10% of grid points have LFC defined. Therefore we do some memory-optimization
	// here: instead of allocating and processing whole grids with 90% missing data,
	// we 'compact' the data with a bitmap.
	//
	// The LFC pressure field is read and each missing grid point is marked with a '1'.
	// The effective grid size in the processing phase is then the number of ones in the
	// bitmap field. Every time a a new field is read from database, it is processed so
	// that grid points who's bitmap value is zero are removed.
	//
	// At the end of processing the grid is again 'expanded' so that the grid point values
	// in the compact grid are moved to correct locations.

	std::vector<unsigned char> bitmap(T.size(), 1);

	// No LFC --> No CAPE

	for (size_t i = 0; i < P.size(); i++)
	{
		if (IsMissing(P[i]))
		{
			bitmap[i] = 0;
		}
	}

	const size_t N = count(bitmap.begin(), bitmap.end(), 1);
	const size_t NB = bitmap.size();

	// 'bitmap' variable is immutable; 'found' variable gets filled up with
	// ones as processing proceeds

	std::vector<unsigned char> found(N, 0);

	logger log("CAPEGPU");

	log.Info("Compacting data with a ratio of " + std::to_string(static_cast<double>(N) / NB) + " (smaller is better)");

	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	float* d_CAPE = 0;
	float* d_CAPE1040 = 0;
	float* d_CAPE3km = 0;
	float* d_ELT = 0;
	float* d_ELP = 0;
	float* d_ELZ = 0;
	float* d_LastELT = 0;
	float* d_LastELP = 0;
	float* d_LastELZ = 0;
	float* d_prevTparcel = 0;
	float* d_Tparcel = 0;
	float* d_LFCT = 0;
	float* d_LFCP = 0;
	float* d_origLFCT = 0;
	float* d_origLFCP = 0;
	float* d_prevZenv = 0;
	float* d_prevPenv = 0;
	float* d_prevTenv = 0;
	float* d_Zenv = 0;
	float* d_Penv = 0;
	float* d_Tenv = 0;
	float* d_origZenv = 0;
	float* d_origPenv = 0;
	float* d_origTenv = 0;

	unsigned char* d_found = 0;
	unsigned char* d_bitmap = 0;

	CUDA_CHECK(hipMalloc((float**)&d_CAPE, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_CAPE1040, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_CAPE3km, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ELP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ELT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_ELZ, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastELP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastELT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LastELZ, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTparcel, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevZenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevTenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_prevPenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Zenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Tenv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_Penv, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_origZenv, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_origTenv, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_origPenv, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_LFCT, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_LFCP, sizeof(float) * N));
	CUDA_CHECK(hipMalloc((float**)&d_origLFCT, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((float**)&d_origLFCP, sizeof(float) * NB));
	CUDA_CHECK(hipMalloc((unsigned char**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((unsigned char**)&d_bitmap, sizeof(unsigned char) * NB));

	CUDA_CHECK(hipMemcpyAsync(d_origLFCP, P.data(), sizeof(float) * NB, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_origLFCT, T.data(), sizeof(float) * NB, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_bitmap, bitmap.data(), sizeof(unsigned char) * NB, hipMemcpyHostToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_LFCT, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_bitmap, bitmap.data(), sizeof(unsigned char) * NB, hipMemcpyHostToDevice, stream));

	auto bitmapHot = [] __device__(const unsigned char& u) { return u == 1; };

	// https://thrust.github.io/doc/group__stream__compaction.html#ga36d9d6ed8e17b442c1fd8dc40bd515d5
	thrust::copy_if(thrust::cuda::par.on(stream), d_origLFCT, d_origLFCT + NB, d_bitmap, d_LFCT, bitmapHot);
	thrust::copy_if(thrust::cuda::par.on(stream), d_origLFCP, d_origLFCP + NB, d_bitmap, d_LFCP, bitmapHot);

	InitializeArray<unsigned char>(d_found, 0., N, stream);
	InitializeArray<float>(d_CAPE, 0., N, stream);
	InitializeArray<float>(d_CAPE1040, 0., N, stream);
	InitializeArray<float>(d_CAPE3km, 0., N, stream);

	InitializeArray<float>(d_ELP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_ELT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_ELZ, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastELP, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastELT, himan::MissingFloat(), N, stream);
	InitializeArray<float>(d_LastELZ, himan::MissingFloat(), N, stream);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_origLFCT));
	CUDA_CHECK(hipFree(d_origLFCP));

	// For each grid point find the hybrid level that's below LFC and then pick the lowest level
	// among all grid points

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P), conf->TargetGeomName());

	level curLevel = levels.first;

	auto prevZenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, ZParam, myTargetInfo->ForecastType());
	auto prevTenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
	auto prevPenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());

	// "orig" variables are just as a temporary placeholder
	hc::PrepareInfo(prevZenvInfo, d_origZenv, stream, conf->UseCacheForReads());
	hc::PrepareInfo(prevPenvInfo, d_origPenv, stream, conf->UseCacheForReads());
	hc::PrepareInfo(prevTenvInfo, d_origTenv, stream, conf->UseCacheForReads());

	thrust::copy_if(thrust::cuda::par.on(stream), d_origZenv, d_origZenv + NB, d_bitmap, d_prevZenv, bitmapHot);
	thrust::copy_if(thrust::cuda::par.on(stream), d_origPenv, d_origPenv + NB, d_bitmap, d_prevPenv, bitmapHot);
	thrust::copy_if(thrust::cuda::par.on(stream), d_origTenv, d_origTenv + NB, d_bitmap, d_prevTenv, bitmapHot);

	if (cape_cuda::itsUseVirtualTemperature)
	{
		VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_prevTenv, d_prevPenv, N);
	}

	curLevel.Value(curLevel.Value());

	auto stopLevel = h->LevelForHeight(myTargetInfo->Producer(), 50., conf->TargetGeomName());
	auto hPa450 = h->LevelForHeight(myTargetInfo->Producer(), 450., conf->TargetGeomName());

	thrust::device_ptr<unsigned char> dt_found = thrust::device_pointer_cast(d_found);

	std::shared_ptr<info<float>> PenvInfo, TenvInfo, ZenvInfo;

	while (curLevel.Value() > stopLevel.first.Value())
	{
		PenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, PParam, myTargetInfo->ForecastType());
		TenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, TParam, myTargetInfo->ForecastType());
		ZenvInfo = hc::Fetch<float>(conf, myTargetInfo->Time(), curLevel, ZParam, myTargetInfo->ForecastType());

		if (!PenvInfo || !TenvInfo || !ZenvInfo)
		{
			break;
		}

		hc::PrepareInfo<float>(ZenvInfo, d_origZenv, stream, conf->UseCacheForReads());
		hc::PrepareInfo<float>(PenvInfo, d_origPenv, stream, conf->UseCacheForReads());
		hc::PrepareInfo<float>(TenvInfo, d_origTenv, stream, conf->UseCacheForReads());

		thrust::copy_if(thrust::cuda::par.on(stream), d_origZenv, d_origZenv + NB, d_bitmap, d_Zenv, bitmapHot);
		thrust::copy_if(thrust::cuda::par.on(stream), d_origPenv, d_origPenv + NB, d_bitmap, d_Penv, bitmapHot);
		thrust::copy_if(thrust::cuda::par.on(stream), d_origTenv, d_origTenv + NB, d_bitmap, d_Tenv, bitmapHot);

		if (cape_cuda::itsUseVirtualTemperature)
		{
			VirtualTemperatureKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, N);
		}

		MoistLiftKernel<<<gridSize, blockSize, 0, stream>>>(d_LFCT, d_LFCP, d_Penv, d_Tparcel, N);

		CAPEKernel<<<gridSize, blockSize, 0, stream>>>(d_Tenv, d_Penv, d_Zenv, d_prevTenv, d_prevPenv, d_prevZenv,
		                                               d_Tparcel, d_prevTparcel, d_LFCT, d_LFCP, d_CAPE, d_CAPE1040,
		                                               d_CAPE3km, d_ELT, d_ELP, d_ELZ, d_LastELT, d_LastELP, d_LastELZ,
		                                               d_found, curLevel.Value(), hPa450.first.Value(), N);

		size_t foundCount = thrust::count(thrust::cuda::par.on(stream), dt_found, dt_found + N, 1);
		CUDA_CHECK(hipStreamSynchronize(stream));

		if (foundCount == N)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevZenv, d_Zenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTenv, d_Tenv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevPenv, d_Penv, sizeof(float) * N, hipMemcpyDeviceToDevice, stream));

		curLevel.Value(curLevel.Value() - 1);
	}

	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_bitmap));
	CUDA_CHECK(hipFree(d_prevTenv));
	CUDA_CHECK(hipFree(d_prevPenv));
	CUDA_CHECK(hipFree(d_prevZenv));
	CUDA_CHECK(hipFree(d_origTenv));
	CUDA_CHECK(hipFree(d_origPenv));
	CUDA_CHECK(hipFree(d_origZenv));

	CapELValuesKernel<<<gridSize, blockSize, 0, stream>>>(d_CAPE, d_ELT, d_ELP, d_ELZ, d_LastELT, d_LastELP, d_LastELZ,
	                                                      d_Tenv, d_Penv, d_Zenv, N);

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_Tenv));
	CUDA_CHECK(hipFree(d_Penv));
	CUDA_CHECK(hipFree(d_Zenv));

	std::vector<float> CAPE(NB, 0);
	std::vector<float> CAPE1040(NB, 0);
	std::vector<float> CAPE3km(NB, 0);
	std::vector<float> ELT(NB, himan::MissingFloat());
	std::vector<float> ELP(NB, himan::MissingFloat());
	std::vector<float> ELZ(NB, himan::MissingFloat());
	std::vector<float> LastELT(NB, himan::MissingFloat());
	std::vector<float> LastELP(NB, himan::MissingFloat());
	std::vector<float> LastELZ(NB, himan::MissingFloat());

	// intermediate containers where data is copied before expansion
	std::vector<float> bm_CAPE(N);
	std::vector<float> bm_CAPE1040(N);
	std::vector<float> bm_CAPE3km(N);
	std::vector<float> bm_ELT(N);
	std::vector<float> bm_ELP(N);
	std::vector<float> bm_ELZ(N);
	std::vector<float> bm_LastELT(N);
	std::vector<float> bm_LastELP(N);
	std::vector<float> bm_LastELZ(N);

	CUDA_CHECK(hipMemcpyAsync(bm_CAPE.data(), d_CAPE, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_CAPE1040.data(), d_CAPE1040, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_CAPE3km.data(), d_CAPE3km, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_ELT.data(), d_ELT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_ELP.data(), d_ELP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_ELZ.data(), d_ELZ, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_LastELT.data(), d_LastELT, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_LastELP.data(), d_LastELP, sizeof(float) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(bm_LastELZ.data(), d_LastELZ, sizeof(float) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_CAPE));
	CUDA_CHECK(hipFree(d_CAPE1040));
	CUDA_CHECK(hipFree(d_CAPE3km));
	CUDA_CHECK(hipFree(d_ELT));
	CUDA_CHECK(hipFree(d_ELP));
	CUDA_CHECK(hipFree(d_ELZ));
	CUDA_CHECK(hipFree(d_LastELT));
	CUDA_CHECK(hipFree(d_LastELP));
	CUDA_CHECK(hipFree(d_LastELZ));

	// "expand" data again with bitmap

	for (size_t i = 0, j = 0; i < NB; i++)
	{
		if (bitmap[i])
		{
			CAPE[i] = bm_CAPE[j];
			CAPE1040[i] = bm_CAPE1040[j];
			CAPE3km[i] = bm_CAPE3km[j];
			ELT[i] = bm_ELT[j];
			ELP[i] = bm_ELP[j];
			ELZ[i] = bm_ELZ[j];
			LastELT[i] = bm_LastELT[j];
			LastELP[i] = bm_LastELP[j];
			LastELZ[i] = bm_LastELZ[j];

			j++;
		}
	}

	return make_tuple(ELT, ELP, ELZ, LastELT, LastELP, LastELZ, CAPE, CAPE1040, CAPE3km);
}
