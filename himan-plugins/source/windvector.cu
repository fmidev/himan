#include "hip/hip_runtime.h"
#include "cuda_plugin_helper.h"
#include "interpolate.h"
#include "plugin_factory.h"
#include "windvector.cuh"
#include <ogr_spatialref.h>

#define HIMAN_AUXILIARY_INCLUDE

#include "cache.h"

#undef HIMAN_AUXILIARY_INCLUDE

static std::map<size_t, double*> longitudeCache;
static std::mutex cacheMutex;

/*
 * Calculate results. At this point it as assumed that U and V are in correct form.
 */

__global__ void Calculate(const float* __restrict__ d_u, const float* __restrict__ d_v, float* __restrict__ d_speed,
                          float* __restrict__ d_dir, himan::plugin::HPWindVectorTargetType targetType, size_t N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		const float U = d_u[idx];
		const float V = d_v[idx];

		d_speed[idx] = __fsqrt_rn(U * U + V * V);

		if (targetType != himan::plugin::kGust)
		{
			int offset = 180;

			if (targetType == himan::plugin::kSea || targetType == himan::plugin::kIce)
			{
				offset = 0;
			}

			float dir = himan::constants::kRad * atan2(U, V) + offset;

			// modulo operator is supposedly slow on cuda ?

			/*
			 * quote:
			 *
			 * Integer division and modulo operation are costly: tens of instructions on devices of
			 * compute capability 1.x, below 20 instructions on devices of compute capability 2.x and
			 * higher.
			 */

			// reduce the angle
			while (dir > 360)
			{
				dir -= 360;
			}

			// force it to be the positive remainder, so that 0 <= dir < 360

			while (dir < 0)
			{
				dir += 360;
			}

			d_dir[idx] = round(dir);
		}
	}
}

void himan::plugin::windvector_cuda::RunCuda(std::shared_ptr<const plugin_configuration> conf,
                                             std::shared_ptr<info<float>> myTargetInfo, const param& UParam,
                                             const param& VParam, HPWindVectorTargetType itsTargetType)
{
	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	// Allocate device arrays

	float* d_u = 0;
	float* d_v = 0;
	float* d_speed = 0;
	float* d_dir = 0;

	// Allocate memory on device
	const size_t N = myTargetInfo->SizeLocations();

	const size_t memsize = N * sizeof(float);

	// Fetch U & V, unpack to device, do not copy to host

	auto UInfo =
	    cuda::Fetch<float>(conf, myTargetInfo->Time(), myTargetInfo->Level(), UParam, myTargetInfo->ForecastType());
	auto VInfo =
	    cuda::Fetch<float>(conf, myTargetInfo->Time(), myTargetInfo->Level(), VParam, myTargetInfo->ForecastType());

	if (!UInfo || !VInfo)
	{
		return;
	}

	CUDA_CHECK(hipMalloc((void**)&d_u, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_v, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_speed, memsize));

	if (itsTargetType != kGust)
	{
		CUDA_CHECK(hipMalloc((void**)&d_dir, memsize));
	}

	cuda::Unpack(UInfo, stream, d_u);
	cuda::Unpack(VInfo, stream, d_v);

	// Rotate components; data already at device memory

	if (UInfo->Grid()->UVRelativeToGrid())
	{
		double* d_lon = windvector_cuda::CacheLongitudeCoordinates(UInfo->Grid().get(), stream);

		latitude_longitude_grid dummy(kBottomLeft, point(), point(), 0, 0, earth_shape<double>());

		himan::interpolate::RotateVectorComponentsGPU(UInfo->Grid().get(), &dummy, UInfo->Data(), VInfo->Data(), stream,
		                                              d_u, d_v, d_lon);
		CUDA_CHECK(hipStreamSynchronize(stream));

		UInfo->Grid()->UVRelativeToGrid(false);
		VInfo->Grid()->UVRelativeToGrid(false);
	}

	// Copy to host

	CUDA_CHECK(hipMemcpyAsync(UInfo->Data().ValuesAsPOD(), d_u, memsize, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(VInfo->Data().ValuesAsPOD(), d_v, memsize, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	// And finally insert to cache

	if (conf->UseCacheForReads())
	{
		auto c = GET_PLUGIN(cache);
		c->Insert(UInfo);
		c->Insert(VInfo);
	}

	if (myTargetInfo->Level().Type() == kHybrid)
	{
		const size_t paramIndex = myTargetInfo->Index<param>();

		for (myTargetInfo->Reset<param>(); myTargetInfo->Next<param>();)
		{
			myTargetInfo->Set<level>(UInfo->Level());
		}

		myTargetInfo->Index<param>(paramIndex);
	}

	// dims

	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	Calculate<<<gridSize, blockSize, 0, stream>>>(d_u, d_v, d_speed, d_dir, itsTargetType, N);

	// block until the stream has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	myTargetInfo->Index<param>(0);

	cuda::ReleaseInfo(myTargetInfo, d_speed, stream);

	if (itsTargetType != kGust)
	{
		myTargetInfo->Index<param>(1);
		cuda::ReleaseInfo(myTargetInfo, d_dir, stream);
	}

	CUDA_CHECK(hipStreamSynchronize(stream));

	// Free device memory

	CUDA_CHECK(hipFree(d_u));
	CUDA_CHECK(hipFree(d_v));
	CUDA_CHECK(hipFree(d_speed));

	if (d_dir)
	{
		CUDA_CHECK(hipFree(d_dir));
	}

	if (itsTargetType == kGust)
	{
		// Check aggregation period from source file
		const auto agg = UInfo->Param().Aggregation();
		if (agg.Type() != kUnknownAggregationType)
		{
			auto& par = myTargetInfo->Param();
			par.Aggregation(agg);
		}
	}

	CUDA_CHECK(hipStreamDestroy(stream));
}

double* himan::plugin::windvector_cuda::CacheLongitudeCoordinates(const himan::grid* g, hipStream_t& stream)
{
	const size_t hash = g->Hash();
	himan::logger log("windvector_cuda");

	if (longitudeCache.find(hash) == longitudeCache.end())
	{
		std::lock_guard<std::mutex> lock(cacheMutex);

		if (longitudeCache.find(hash) == longitudeCache.end())
		{
			double* d_lon = nullptr;
			std::vector<double> lon(g->Size());

			CUDA_CHECK(hipMalloc((void**)&d_lon, g->Size() * sizeof(double)));

			for (size_t i = 0; i < g->Size(); i++)
			{
				lon[i] = g->LatLon(i).X();
			}
			CUDA_CHECK(hipMemcpyAsync(d_lon, lon.data(), g->Size() * sizeof(double), hipMemcpyHostToDevice, stream));

			longitudeCache.emplace(hash, d_lon);
			log.Trace("Add longitude cache for " + std::to_string(hash));
			CUDA_CHECK(hipStreamSynchronize(stream));
		}
		else
		{
			log.Trace("Found longitude cache for " + std::to_string(hash));
		}
	}
	else
	{
		log.Trace("Found longitude cache for " + std::to_string(hash));
	}

	return longitudeCache[hash];
}

void himan::plugin::windvector_cuda::FreeLongitudeCache()
{
	himan::logger log("windvector_cuda");
	std::lock_guard<std::mutex> lock(cacheMutex);

	for (auto& p : longitudeCache)
	{
		if (p.second == nullptr)
		{
			continue;
		}
		CUDA_CHECK(hipFree(p.second));
		log.Trace("Cleared longitude cache for " + std::to_string(p.first));
	}

	longitudeCache.clear();
}
